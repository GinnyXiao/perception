
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DSIZE 10
#define nTPB 256

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

typedef union  {
  float floats[2];                 // floats[0] = lowest
  int ints[4];                     // ints[1] = lowIdx
  unsigned long long int ulong;    // for atomic update
} my_atomics;

__device__ my_atomics test;


__device__ unsigned long long int my_atomicMin(unsigned long long int* address, float val1, int val2,int row)
{
	int idx = (blockDim.x * blockIdx.x) + threadIdx.x;    
	my_atomics loc, loctest;
  loc.floats[row] = val1;
  loc.ints[row+2] = val2;
  loctest.ulong = *address;
  while (loctest.floats[row] >  val1){
    loctest.ulong = atomicCAS(address, loctest.ulong,  loc.ulong);
  }
  return loctest.ulong;
}


__global__ void min_test(const float* data)
{

    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (idx < DSIZE)
      my_atomicMin(&(test.ulong), data[idx],idx,0);
    if (idx >= DSIZE && idx < 2*DSIZE)
      my_atomicMin(&(test.ulong), data[idx],idx,1);
}

int main() {

  float *d_data, *h_data;
  my_atomics my_init;
  my_init.floats[0] = 10.0f;
  my_init.ints[1] = DSIZE;

  h_data = (float *)malloc(2*DSIZE * sizeof(float));
  if (h_data == 0) {printf("malloc fail\n"); return 1;}
  hipMalloc((void **)&d_data, DSIZE*2 * sizeof(float));
  cudaCheckErrors("cm1 fail");
  // create random floats between 0 and 1
  for (int i = 0; i < DSIZE; i++) h_data[i] = rand()/(float)RAND_MAX;
  hipMemcpy(d_data, h_data, 2*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("cmcp1 fail");
  hipMemcpyToSymbol(HIP_SYMBOL(test), &(my_init.ulong), sizeof(unsigned long long int));
  cudaCheckErrors("cmcp2 fail");
  min_test<<<(DSIZE+nTPB-1)/nTPB, nTPB>>>(d_data);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel fail");

  hipMemcpyFromSymbol(&(my_init.ulong), HIP_SYMBOL(test), sizeof(unsigned long long int));
  cudaCheckErrors("cmcp3 fail");

  printf("device min result = %f\n", my_init.floats[0]);
  printf("device idx result = %d\n", my_init.ints[2]);
  printf("device min result = %f\n", my_init.floats[1]);
  printf("device idx result = %d\n", my_init.ints[3]);
  float host_val = 10.0f;
  int host_idx = DSIZE;
  for (int i=0; i<DSIZE; i++)
    if (h_data[i] < host_val){
      host_val = h_data[i];
      host_idx = i;
      }

  printf("host min result = %f\n", host_val);
  printf("host idx result = %d\n", host_idx);
  return 0;
}
