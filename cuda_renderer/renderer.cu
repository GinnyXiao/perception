#include "hip/hip_runtime.h"
#include "cuda_renderer/renderer.h"
// #include <math.h> 
#include "hip/hip_fp16.h"
// #include <numeric> 
#define SQR(x) ((x)*(x))
#define POW2(x) SQR(x)
#define POW3(x) ((x)*(x)*(x))
#define POW4(x) (POW2(x)*POW2(x))
#define POW7(x) (POW3(x)*POW3(x)*(x))
#define DegToRad(x) ((x)*M_PI/180)
#define RadToDeg(x) ((x)/M_PI*180)

namespace cuda_renderer {
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if(err!=hipSuccess)
    {
        fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);
    }
}
#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


template<typename T>
device_vector_holder<T>::~device_vector_holder(){
    __free();
}

template<typename T>
void device_vector_holder<T>::__free(){
    if(valid){
        hipFree(__gpu_memory);
        valid = false;
        __size = 0;
    }
}

template<typename T>
device_vector_holder<T>::device_vector_holder(size_t size_, T init)
{
    __malloc(size_);
    thrust::fill(begin_thr(), end_thr(), init);
}

template<typename T>
void device_vector_holder<T>::__malloc(size_t size_){
    if(valid) __free();
    hipMalloc((void**)&__gpu_memory, size_ * sizeof(T));
    __size = size_;
    valid = true;
}

template<typename T>
device_vector_holder<T>::device_vector_holder(size_t size_){
    __malloc(size_);
}

template class device_vector_holder<int>;

void print_cuda_memory_usage(){
    // show memory usage of GPU

    size_t free_byte ;
    size_t total_byte ;
    auto cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
        used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

struct max2zero_functor{

    max2zero_functor(){}

    __host__ __device__
    int32_t operator()(const int32_t& x) const
    {
      return (x==INT_MAX)? 0: x;
    }
};

__device__ void rgb2lab(uint8_t rr,uint8_t gg, uint8_t bbb, float* lab){
    double r = rr / 255.0;
    double g = gg / 255.0;
    double b = bbb / 255.0;
    double x;
    double y;
    double z;
    r = ((r > 0.04045) ? pow((r + 0.055) / 1.055, 2.4) : (r / 12.92)) * 100.0;
    g = ((g > 0.04045) ? pow((g + 0.055) / 1.055, 2.4) : (g / 12.92)) * 100.0;
    b = ((b > 0.04045) ? pow((b + 0.055) / 1.055, 2.4) : (b / 12.92)) * 100.0;

    x = r*0.4124564 + g*0.3575761 + b*0.1804375;
    y = r*0.2126729 + g*0.7151522 + b*0.0721750;
    z = r*0.0193339 + g*0.1191920 + b*0.9503041;

    x = x / 95.047;
    y = y / 100.00;
    z = z / 108.883;

    x = (x > 0.008856) ? cbrt(x) : (7.787 * x + 16.0 / 116.0);
    y = (y > 0.008856) ? cbrt(y) : (7.787 * y + 16.0 / 116.0);
    z = (z > 0.008856) ? cbrt(z) : (7.787 * z + 16.0 / 116.0);
    float l,a,bb;

    l = (116.0 * y) - 16;
    a = 500 * (x - y);
    bb = 200 * (y - z);

    lab[0] = l;
    lab[1] = a;
    lab[2] = bb;
}
__device__ double color_distance(float l1,float a1,float b1,
                      float l2,float a2,float b2){
    double eps = 1e-5;
    double c1 = sqrtf(SQR(a1) + SQR(b1));
    double c2 = sqrtf(SQR(a2) + SQR(b2));
    double meanC = (c1 + c2) / 2.0;
    double meanC7 = POW7(meanC);

    double g = 0.5*(1 - sqrtf(meanC7 / (meanC7 + 6103515625.))); // 0.5*(1-sqrt(meanC^7/(meanC^7+25^7)))
    double a1p = a1 * (1 + g);
    double a2p = a2 * (1 + g);

    c1 = sqrtf(SQR(a1p) + SQR(b1));
    c2 = sqrtf(SQR(a2p) + SQR(b2));
    double h1 = fmodf(atan2f(b1, a1p) + 2*M_PI, 2*M_PI);
    double h2 = fmodf(atan2f(b2, a2p) + 2*M_PI, 2*M_PI);

    // compute deltaL, deltaC, deltaH
    double deltaL = l2 - l1;
    double deltaC = c2 - c1;
    double deltah;

    if (c1*c2 < eps) {
        deltah = 0;
    }
    if (std::abs(h2 - h1) <= M_PI) {
        deltah = h2 - h1;
    }
    else if (h2 > h1) {
        deltah = h2 - h1 - 2* M_PI;
    }
    else {
        deltah = h2 - h1 + 2 * M_PI;
    }

    double deltaH = 2 * sqrtf(c1*c2)*sinf(deltah / 2);

    // calculate CIEDE2000
    double meanL = (l1 + l2) / 2;
    meanC = (c1 + c2) / 2.0;
    meanC7 = POW7(meanC);
    double meanH;

    if (c1*c2 < eps) {
        meanH = h1 + h2;
    }
    if (std::abs(h1 - h2) <= M_PI + eps) {
        meanH = (h1 + h2) / 2;
    }
    else if (h1 + h2 < 2*M_PI) {
        meanH = (h1 + h2 + 2*M_PI) / 2;
    }
    else {
        meanH = (h1 + h2 - 2*M_PI) / 2;
    }

    double T = 1
        - 0.17*cosf(meanH - DegToRad(30))
        + 0.24*cosf(2 * meanH)
        + 0.32*cosf(3 * meanH + DegToRad(6))
        - 0.2*cosf(4 * meanH - DegToRad(63));
    double sl = 1 + (0.015*SQR(meanL - 50)) / sqrtf(20 + SQR(meanL - 50));
    double sc = 1 + 0.045*meanC;
    double sh = 1 + 0.015*meanC*T;
    double rc = 2 * sqrtf(meanC7 / (meanC7 + 6103515625.));
    double rt = -sinf(DegToRad(60 * expf(-SQR((RadToDeg(meanH) - 275) / 25)))) * rc;

    double cur_dist = sqrtf(SQR(deltaL / sl) + SQR(deltaC / sc) + SQR(deltaH / sh) + rt * deltaC / sc * deltaH / sh);
    return cur_dist;
}
__device__
void rasterization_with_source(const Model::Triangle dev_tri, Model::float3 last_row,
                                        int32_t* depth_entry, size_t width, size_t height,
                                        const Model::ROI roi, 
                                        uint8_t* red_entry,uint8_t* green_entry,uint8_t* blue_entry,
                                        int32_t* source_depth_entry,
                                        uint8_t* source_red_entry,uint8_t* source_green_entry,uint8_t* source_blue_entry,
                                        int* pose_occluded_entry,
                                        int32_t* lock_entry,
                                        int* pose_occluded_other_entry,
                                        float* pose_clutter_points_entry,
                                        float* pose_total_points_entry) {
                                        // float* l_entry,float* a_entry,float* b_entry){
    // refer to tiny renderer
    // https://github.com/ssloy/tinyrenderer/blob/master/our_gl.cpp
    float pts2[3][2];

    // viewport transform(0, 0, width, height)
    pts2[0][0] = dev_tri.v0.x/last_row.x*width/2.0f+width/2.0f; pts2[0][1] = dev_tri.v0.y/last_row.x*height/2.0f+height/2.0f;
    pts2[1][0] = dev_tri.v1.x/last_row.y*width/2.0f+width/2.0f; pts2[1][1] = dev_tri.v1.y/last_row.y*height/2.0f+height/2.0f;
    pts2[2][0] = dev_tri.v2.x/last_row.z*width/2.0f+width/2.0f; pts2[2][1] = dev_tri.v2.y/last_row.z*height/2.0f+height/2.0f;

    float bboxmin[2] = {FLT_MAX,  FLT_MAX};
    float bboxmax[2] = {-FLT_MAX, -FLT_MAX};

    float clamp_max[2] = {float(width-1), float(height-1)};
    float clamp_min[2] = {0, 0};

    size_t real_width = width;
    if(roi.width > 0 && roi.height > 0){  // depth will be flipped
        clamp_min[0] = roi.x;
        clamp_min[1] = height-1 - (roi.y + roi.height - 1);
        clamp_max[0] = (roi.x + roi.width) - 1;
        clamp_max[1] = height-1 - roi.y;
        real_width = roi.width;
    }


    for (int i=0; i<3; i++) {
        for (int j=0; j<2; j++) {
            bboxmin[j] = std__max(clamp_min[j], std__min(bboxmin[j], pts2[i][j]));
            bboxmax[j] = std__min(clamp_max[j], std__max(bboxmax[j], pts2[i][j]));
        }
    }

    size_t P[2];
    for(P[1] = size_t(bboxmin[1]+0.5f); P[1]<=bboxmax[1]; P[1] += 1){
        for(P[0] = size_t(bboxmin[0]+0.5f); P[0]<=bboxmax[0]; P[0] += 1){
            Model::float3 bc_screen  = barycentric(pts2[0], pts2[1], pts2[2], P);

            if (bc_screen.x<-0.0f || bc_screen.y<-0.0f || bc_screen.z<-0.0f ||
                    bc_screen.x>1.0f || bc_screen.y>1.0f || bc_screen.z>1.0f ) continue;

            Model::float3 bc_over_z = {bc_screen.x/last_row.x, bc_screen.y/last_row.y, bc_screen.z/last_row.z};

            // refer to https://en.wikibooks.org/wiki/Cg_Programming/Rasterization, Perspectively Correct Interpolation
//            float frag_depth = (dev_tri.v0.z * bc_over_z.x + dev_tri.v1.z * bc_over_z.y + dev_tri.v2.z * bc_over_z.z)
//                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

            // this seems better
            float frag_depth = (bc_screen.x + bc_screen.y + bc_screen.z)
                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

            size_t x_to_write = (P[0] + roi.x);
            size_t y_to_write = (height-1 - P[1] - roi.y);
            int32_t curr_depth = int32_t(frag_depth/**1000*/ + 0.5f);
            // printf("x:%d, y:%d, depth:%d\n", x_to_write, y_to_write, curr_depth);
            int32_t& depth_to_write = depth_entry[x_to_write+y_to_write*real_width];
            int32_t& source_depth = source_depth_entry[x_to_write+y_to_write*real_width];
            uint8_t source_red = source_red_entry[x_to_write+y_to_write*real_width];
            uint8_t source_green = source_green_entry[x_to_write+y_to_write*real_width];
            uint8_t source_blue = source_blue_entry[x_to_write+y_to_write*real_width];

            // if(depth_to_write > curr_depth){
            //     red_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v0);
            //     green_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v1);
            //     blue_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v2);
            // }
            // atomicMin(&depth_to_write, curr_depth);
            bool wait = true;
            while(wait){
                if(0 == atomicExch(&lock_entry[x_to_write+y_to_write*real_width], 1)){
                    if(curr_depth < depth_entry[x_to_write+y_to_write*real_width]){
                        depth_entry[x_to_write+y_to_write*real_width] = curr_depth;
                        red_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v0);
                        green_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v1);
                        blue_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v2);
                    }
                    lock_entry[x_to_write+y_to_write*real_width] = 0;
                    wait = false;
                }
            }
            int32_t& new_depth = depth_entry[x_to_write+y_to_write*real_width];
            if(new_depth > source_depth && source_depth > 0){
                // when we are rendering at x,y where source pixel is also present at depth closer to camera
                // valid condition as source occludes render
                // if (false)
                // {
                //     // add source pixels
                //     red_entry[x_to_write+y_to_write*real_width] = source_red;
                //     green_entry[x_to_write+y_to_write*real_width] = source_green;
                //     blue_entry[x_to_write+y_to_write*real_width] = source_blue;
                //     atomicMin(&new_depth, source_depth);
                // }
                // else
                // {
                    // add black
                    red_entry[x_to_write+y_to_write*real_width] = 0;
                    green_entry[x_to_write+y_to_write*real_width] = 0;
                    blue_entry[x_to_write+y_to_write*real_width] = 0;
                    atomicMax(&new_depth, INT_MAX);
                    atomicOr(pose_occluded_other_entry, 1);
                    if (source_depth <=  new_depth - 5)
                    {
                        atomicAdd(pose_clutter_points_entry, 1);
                    }
                // }
            }
            // invalid condition where source pixel is behind and we are rendering a pixel at same x,y with lesser depth 
            else if(new_depth <= source_depth && source_depth > 0){
                // invalid as render occludes source
                atomicOr(pose_occluded_entry, 1);
                // printf("Occlusion\n");
            }
            atomicAdd(pose_total_points_entry, 1);

        }
    }
}
__device__
void rasterization(const Model::Triangle dev_tri, Model::float3 last_row,
                                        int32_t* depth_entry, size_t width, size_t height,
                                        const Model::ROI roi, uint8_t* red_entry,uint8_t* green_entry,uint8_t* blue_entry){
                                        // float* l_entry,float* a_entry,float* b_entry){
    // refer to tiny renderer
    // https://github.com/ssloy/tinyrenderer/blob/master/our_gl.cpp
    float pts2[3][2];

    // viewport transform(0, 0, width, height)
    pts2[0][0] = dev_tri.v0.x/last_row.x*width/2.0f+width/2.0f; pts2[0][1] = dev_tri.v0.y/last_row.x*height/2.0f+height/2.0f;
    pts2[1][0] = dev_tri.v1.x/last_row.y*width/2.0f+width/2.0f; pts2[1][1] = dev_tri.v1.y/last_row.y*height/2.0f+height/2.0f;
    pts2[2][0] = dev_tri.v2.x/last_row.z*width/2.0f+width/2.0f; pts2[2][1] = dev_tri.v2.y/last_row.z*height/2.0f+height/2.0f;

    float bboxmin[2] = {FLT_MAX,  FLT_MAX};
    float bboxmax[2] = {-FLT_MAX, -FLT_MAX};

    float clamp_max[2] = {float(width-1), float(height-1)};
    float clamp_min[2] = {0, 0};

    size_t real_width = width;
    if(roi.width > 0 && roi.height > 0){  // depth will be flipped
        clamp_min[0] = roi.x;
        clamp_min[1] = height-1 - (roi.y + roi.height - 1);
        clamp_max[0] = (roi.x + roi.width) - 1;
        clamp_max[1] = height-1 - roi.y;
        real_width = roi.width;
    }


    for (int i=0; i<3; i++) {
        for (int j=0; j<2; j++) {
            bboxmin[j] = std__max(clamp_min[j], std__min(bboxmin[j], pts2[i][j]));
            bboxmax[j] = std__min(clamp_max[j], std__max(bboxmax[j], pts2[i][j]));
        }
    }

    size_t P[2];
    for(P[1] = size_t(bboxmin[1]+0.5f); P[1]<=bboxmax[1]; P[1] += 1){
        for(P[0] = size_t(bboxmin[0]+0.5f); P[0]<=bboxmax[0]; P[0] += 1){
            Model::float3 bc_screen  = barycentric(pts2[0], pts2[1], pts2[2], P);

            if (bc_screen.x<-0.0f || bc_screen.y<-0.0f || bc_screen.z<-0.0f ||
                    bc_screen.x>1.0f || bc_screen.y>1.0f || bc_screen.z>1.0f ) continue;

            Model::float3 bc_over_z = {bc_screen.x/last_row.x, bc_screen.y/last_row.y, bc_screen.z/last_row.z};

            // refer to https://en.wikibooks.org/wiki/Cg_Programming/Rasterization, Perspectively Correct Interpolation
//            float frag_depth = (dev_tri.v0.z * bc_over_z.x + dev_tri.v1.z * bc_over_z.y + dev_tri.v2.z * bc_over_z.z)
//                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

            // this seems better
            float frag_depth = (bc_screen.x + bc_screen.y + bc_screen.z)
                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

            size_t x_to_write = (P[0] + roi.x);
            size_t y_to_write = (height-1 - P[1] - roi.y);

            int32_t depth = int32_t(frag_depth/**1000*/ + 0.5f);
            int32_t& depth_to_write = depth_entry[x_to_write+y_to_write*real_width];

            if(depth_entry[x_to_write+y_to_write*real_width] > depth){
                red_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v0);
                green_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v1);
                blue_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v2);
            }
            atomicMin(&depth_to_write,depth);
        }
    }
}

__global__ void render_triangle(Model::Triangle* device_tris_ptr, size_t device_tris_size,
                                Model::mat4x4* device_poses_ptr, size_t device_poses_size,
                                int32_t* depth_image_vec, size_t width, size_t height, const Model::mat4x4 proj_mat,
                                 const Model::ROI roi,uint8_t* red_image_vec,uint8_t* green_image_vec,uint8_t* blue_image_vec){
                                 // float* l_vec,float* a_vec,float* b_vec){
    size_t pose_i = blockIdx.y;
    size_t tri_i = blockIdx.x*blockDim.x + threadIdx.x;

    if(tri_i>=device_tris_size) return;
//    if(pose_i>=device_poses_size) return;

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
    }

    int32_t* depth_entry = depth_image_vec + pose_i*real_width*real_height; //length: width*height 32bits int
    uint8_t* red_entry = red_image_vec + pose_i*real_width*real_height;
    uint8_t* green_entry = green_image_vec + pose_i*real_width*real_height;
    uint8_t* blue_entry = blue_image_vec + pose_i*real_width*real_height;
    // float* l_entry = l_vec + pose_i*real_width*real_height;
    // float* a_entry = a_vec + pose_i*real_width*real_height;
    // float* b_entry = b_vec + pose_i*real_width*real_height;
    Model::mat4x4* pose_entry = device_poses_ptr + pose_i; // length: 16 32bits float
    Model::Triangle* tri_entry = device_tris_ptr + tri_i; // length: 9 32bits float

    // model transform
    Model::Triangle local_tri = transform_triangle(*tri_entry, *pose_entry);
//    if(normal_functor::is_back(local_tri)) return; //back face culling, need to be disable for not well defined surfaces?

    // assume last column of projection matrix is  0 0 1 0
    Model::float3 last_row = {
        local_tri.v0.z,
        local_tri.v1.z,
        local_tri.v2.z
    };
    // projection transform
    local_tri = transform_triangle(local_tri, proj_mat);

    // rasterization(local_tri, last_row, depth_entry, width, height, roi,red_entry,green_entry,blue_entry,l_entry,a_entry,b_entry);
    rasterization(local_tri, last_row, depth_entry, width, height, roi,red_entry,green_entry,blue_entry);
}
__global__ void render_triangle_multi(
                                Model::Triangle* device_tris_ptr, size_t device_tris_size,
                                Model::mat4x4* device_poses_ptr, size_t device_poses_size,
                                int32_t* depth_image_vec, size_t width, size_t height,
                                int* device_pose_model_map_ptr, int* device_tris_model_count_low_ptr,  
                                int* device_tris_model_count_high_ptr,
                                const Model::mat4x4 proj_mat, const Model::ROI roi,
                                uint8_t* red_image_vec,uint8_t* green_image_vec,uint8_t* blue_image_vec,
                                int32_t* device_source_depth_vec,
                                uint8_t* device_source_red_vec,uint8_t* device_source_green_vec,uint8_t* device_source_blue_vec,
                                int* pose_occluded_vec,
                                int* device_single_result_image,
                                int32_t* lock_int_vec,
                                int* pose_occluded_other_vec,
                                float* pose_clutter_points_vec, 
                                float* pose_total_points_vec) {
    size_t pose_i = blockIdx.y;
    int model_id = device_pose_model_map_ptr[pose_i];
    size_t tri_i = blockIdx.x*blockDim.x + threadIdx.x;

    if(tri_i>=device_tris_size) return;

    if (!(tri_i < device_tris_model_count_high_ptr[model_id] && tri_i >= device_tris_model_count_low_ptr[model_id]))
        return; 

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
    }
    int32_t* depth_entry;
    int32_t* lock_entry;
    uint8_t* red_entry;
    uint8_t* green_entry;
    uint8_t* blue_entry;
    int* pose_occluded_entry;
    int* pose_occluded_other_entry;
    float* pose_clutter_points_entry;
    float* pose_total_points_entry;
    // printf("device_single_result_image:%d\n",device_single_result_image);
    if (*device_single_result_image)
    {
        depth_entry = depth_image_vec; //length: width*height 32bits int
        red_entry = red_image_vec;
        green_entry = green_image_vec;
        blue_entry = blue_image_vec;
        pose_occluded_entry = pose_occluded_vec;
        lock_entry = lock_int_vec;
        pose_occluded_other_entry = pose_occluded_other_vec;
        pose_clutter_points_entry = pose_clutter_points_vec;
        pose_total_points_entry = pose_total_points_vec;
    }
    else
    {
        depth_entry = depth_image_vec + pose_i*real_width*real_height; //length: width*height 32bits int
        lock_entry = lock_int_vec + pose_i*real_width*real_height;
        red_entry = red_image_vec + pose_i*real_width*real_height;
        green_entry = green_image_vec + pose_i*real_width*real_height;
        blue_entry = blue_image_vec + pose_i*real_width*real_height;
        pose_occluded_entry = pose_occluded_vec + pose_i;
        pose_occluded_other_entry = pose_occluded_other_vec + pose_i;
        pose_clutter_points_entry = pose_clutter_points_vec + pose_i;
        pose_total_points_entry = pose_total_points_vec + pose_i;
    }
    

    Model::mat4x4* pose_entry = device_poses_ptr + pose_i; // length: 16 32bits float
    Model::Triangle* tri_entry = device_tris_ptr + tri_i; // length: 9 32bits float

    // model transform
    Model::Triangle local_tri = transform_triangle(*tri_entry, *pose_entry);

    // assume last column of projection matrix is  0 0 1 0
    Model::float3 last_row = {
        local_tri.v0.z,
        local_tri.v1.z,
        local_tri.v2.z
    };
    // projection transform
    local_tri = transform_triangle(local_tri, proj_mat);

    // rasterization(local_tri, last_row, depth_entry, width, height, roi,red_entry,green_entry,blue_entry);
    rasterization_with_source(
        local_tri, last_row, depth_entry, width, height, roi,
        red_entry,green_entry,blue_entry,
        device_source_depth_vec,
        device_source_red_vec, device_source_green_vec, device_source_blue_vec,
        pose_occluded_entry,
        lock_entry,
        pose_occluded_other_entry,
        pose_clutter_points_entry,
        pose_total_points_entry);
}
__global__ void bgr_to_gray_kernel( uint8_t* red_in,uint8_t* green_in,uint8_t* blue_in,
                                    uint8_t* red_ob, uint8_t* green_ob,uint8_t* blue_ob, 
                                    int32_t* output, 
                                    int width,
                                    int height,
                                    int num_rendered)
{
 //2D Index of current thread
    int num = (int)floorf((blockIdx.x * blockDim.x + threadIdx.x)/width);
    const int xIndex = (blockIdx.x * blockDim.x + threadIdx.x)%width;
    // const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int real_distance;
    //Only valid threads perform memory I/O
    if(xIndex == 213 && yIndex == 143 ){
        // printf("bbbb%d\n", num);
    }
    if((xIndex<width) && (yIndex<height))
    {
        //Location of colored pixel in input
        int valid;
        int cur_id = yIndex * width + xIndex+ num*width*height;
        // int input_id = num*width*height+cur_id;
        
        uint8_t red = red_in[cur_id];
        uint8_t green = green_in[cur_id];
        uint8_t blue = blue_in[cur_id];
       // if(red != 0 || green != 0|| blue != 0){
        float l1,a1,b1,l2,a2,b2; 
        if(red ==0 && green == 0 && blue ==0){
            l1  = 0;
            a1  = 0;
            b1  = 0;
            int id_b = yIndex * width + xIndex;
            if(red_ob[id_b]==0 && green_ob[id_b]==0 && blue_ob[id_b]==0){

            }else{
                uint8_t v =0;
                for(int i = -2; i <3;i++){
                    id_b = (yIndex+i) * width + xIndex+i;
                    if(red_ob[id_b]==0 && green_ob[id_b]==0 && blue_ob[id_b]==0){
                        v = 1;
                    }
                }
                if(v == 0){
                    output[cur_id] = 1;
                }
                
            }

        }else{
            output[cur_id] = 1;
            float lab[3];
            rgb2lab(red,green,blue,lab);
            l1  = lab[0];
            a1  = lab[1];
            b1  = lab[2];
            output[cur_id] = 1;

            for(int i = -2; i <3;i++){
                int row = yIndex+i;
                int col = xIndex+i;
                if(row >= 0 && row <height && col >= 0 && col <width){
                    int id = (row) * width + col;
                    uint8_t red2  = red_ob[id];
                    uint8_t green2  = green_ob[id];
                    uint8_t blue2  = blue_ob[id];
                    if(red2 ==0 && green2 == 0 && blue2 ==0){
                    l2  = 0;
                    a2  = 0;
                    b2  = 0;

                    }else{
                        float lab2[3];
                        rgb2lab(red2,green2,blue2,lab2);
                        l2  = lab2[0];
                        a2  = lab2[1];
                        b2  = lab2[2];
                    }
                    double cur_dist=color_distance(l1,a1,b1,l2,a2,b2);
                    if(cur_dist<20){
                        valid = 1;
                        output[cur_id] = 0;

                    }
                         
                        // if(i==0){
                        //     real_distance = cur_dist;
                        // }
                }

            }
        }   
    }
}

std::vector<int> compute_rgb_cost(const std::vector<std::vector<uint8_t>> input,
                                  const std::vector<std::vector<uint8_t>> observed,
                                  size_t height, size_t width,size_t num_rendered) 
{
    //Calculate total number of bytes of input and output image
    // std::cout<<"aaa";
    size_t bytes = input[0].size();
    size_t bytes_ob = observed[0].size();
   
    // //Allocate device memory
    thrust::device_vector<int> d_output(num_rendered*width*height, 0);
    // this contains all images
    thrust::device_vector<uint8_t> d_red_in = input[0];
    thrust::device_vector<uint8_t> d_green_in = input[1];
    thrust::device_vector<uint8_t> d_blue_in = input[2];
    thrust::device_vector<uint8_t> d_red_ob = observed[0];
    thrust::device_vector<uint8_t> d_green_ob = observed[1];
    thrust::device_vector<uint8_t> d_blue_ob = observed[2];
    {

        int32_t* depth_vec = thrust::raw_pointer_cast(d_output.data());
        uint8_t* red_in = thrust::raw_pointer_cast(d_red_in.data());
        uint8_t* green_in = thrust::raw_pointer_cast(d_green_in.data());
        uint8_t* blue_in = thrust::raw_pointer_cast(d_blue_in.data());
        uint8_t* red_ob = thrust::raw_pointer_cast(d_red_ob.data());
        uint8_t* green_ob = thrust::raw_pointer_cast(d_green_ob.data());
        uint8_t* blue_ob = thrust::raw_pointer_cast(d_blue_ob.data());

        dim3 block(16,16);
        dim3 grid((width*num_rendered + block.x - 1)/block.x, (height + block.y - 1)/block.y);
        bgr_to_gray_kernel<<<grid,block>>>(red_in,green_in,blue_in,
                                       red_ob,green_ob,blue_ob,
                                       depth_vec,
                                       width,height,num_rendered);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
    }

   
    std::vector<int> result_depth(num_rendered*width*height);
    {
        thrust::transform(d_output.begin(), d_output.end(),
                          d_output.begin(), max2zero_functor());
        thrust::copy(d_output.begin(), d_output.end(), result_depth.begin());

    }
    
    // //Copy back data from destination device meory to OpenCV output image
    std::vector<int> cost(num_rendered);
    for(int i = 0 ; i < num_rendered; i ++){
        cost[i] = std::accumulate(result_depth.begin()+i*width*height,result_depth.begin()+(i+1)*width*height,0);
    }
    // std::cout<< cost[0] <<"!!!!!!!!!!";
    
    return cost;
}
__global__ void merge_with_source( uint8_t* merged_red,uint8_t* merged_green,uint8_t* merged_blue,
                                    int32_t* merged_depth,
                                    uint8_t* rendered_red, uint8_t* rendered_green,uint8_t* rendered_blue, 
                                    int32_t* rendered_depth,
                                    uint8_t* source_red, uint8_t* source_green,uint8_t* source_blue, 
                                    int32_t* source_depth, 
                                    int width,
                                    int height,
                                    int num_rendered)
{
    //2D Index of current thread
    int n = (int)floorf((blockIdx.x * blockDim.x + threadIdx.x)/width);
    const int x = (blockIdx.x * blockDim.x + threadIdx.x)%width;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= width) return;
    if(y >= height) return;
    uint32_t idx_rendered = n * width * height + x + y*width;
    uint32_t idx_source = x + y*width;

    if (source_depth[idx_source] > 0 && rendered_depth[idx_rendered] == INT_MAX)
    {

    }

}
__global__ void copy_source_to_render(uint8_t* rendered_red, uint8_t* rendered_green,uint8_t* rendered_blue, 
                                    int32_t* rendered_depth,
                                    uint8_t* source_red, uint8_t* source_green,uint8_t* source_blue, 
                                    int32_t* source_depth, 
                                    int width,
                                    int height,
                                    int num_rendered)
{
    //2D Index of current thread
    int n = (int)floorf((blockIdx.x * blockDim.x + threadIdx.x)/width);
    const int x = (blockIdx.x * blockDim.x + threadIdx.x)%width;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= width) return;
    if(y >= height) return;
    uint32_t idx_rendered = n * width * height + x + y*width;
    uint32_t idx_source = x + y*width;

    if (source_depth[idx_source] > 0)
    {
        rendered_red[idx_rendered] = source_red[idx_source];
        rendered_green[idx_rendered] = source_green[idx_source];
        rendered_blue[idx_rendered] = source_blue[idx_source];
        rendered_depth[idx_rendered] = source_depth[idx_source];
    }

}
device_vector_holder<int> render_cuda_multi(
                            const std::vector<Model::Triangle>& tris,
                            const std::vector<Model::mat4x4>& poses,
                            const std::vector<int> pose_model_map,
                            const std::vector<int> tris_model_count,
                            size_t width, size_t height, const Model::mat4x4& proj_mat,
                            const std::vector<int32_t>& source_depth,
                            const std::vector<std::vector<uint8_t>>& source_color,
                            std::vector<int32_t>& result_depth, 
                            std::vector<std::vector<uint8_t>>& result_color,
                            std::vector<int>& pose_occluded,
                            int single_result_image,
                            std::vector<int>& pose_occluded_other,
                            std::vector<float>& clutter_cost) {

    // Create device inputs
    int* device_single_result_image;
    hipMalloc((void**)&device_single_result_image, sizeof(int));
    hipMemcpy(device_single_result_image, &single_result_image, sizeof(int), hipMemcpyHostToDevice);
    int num_images;
    if (single_result_image)
    {
        num_images = 1;
    }
    else
    {
        num_images = poses.size();
    }
    const Model::ROI roi= {0, 0, 0, 0};
    const size_t threadsPerBlock = 256;
    // std::cout <<tris[0].color.v1;
    thrust::device_vector<Model::Triangle> device_tris = tris;
    thrust::device_vector<Model::mat4x4> device_poses = poses;
    thrust::device_vector<int> device_tris_model_count_low = tris_model_count;
    thrust::device_vector<int> device_tris_model_count_high = tris_model_count;
    thrust::device_vector<int> device_pose_model_map = pose_model_map;

    thrust::device_vector<int32_t> device_source_depth = source_depth;
    thrust::device_vector<uint8_t> device_source_color_red = source_color[0];
    thrust::device_vector<uint8_t> device_source_color_green = source_color[1];
    thrust::device_vector<uint8_t> device_source_color_blue = source_color[2];
    // thrust::copy(
    //     device_tris_model_count.begin(),
    //     device_tris_model_count.end(), 
    //     std::ostream_iterator<int>(std::cout, " ")
    // );
    thrust::exclusive_scan(
        device_tris_model_count_low.begin(), device_tris_model_count_low.end(), 
        device_tris_model_count_low.begin(), 0
    ); // in-place scan
    thrust::inclusive_scan(
        device_tris_model_count_high.begin(), device_tris_model_count_high.end(), 
        device_tris_model_count_high.begin()
    ); // in-place scan
    // thrust::copy(
    //     device_tris_model_count_low.begin(),
    //     device_tris_model_count_low.end(), 
    //     std::ostream_iterator<int>(std::cout, " ")
    // );
    // printf("\n");
    // thrust::copy(
    //     device_tris_model_count_high.begin(),
    //     device_tris_model_count_high.end(), 
    //     std::ostream_iterator<int>(std::cout, " ")
    // );
    // printf("\n");
    // thrust::copy(
    //     device_pose_model_map.begin(),
    //     device_pose_model_map.end(), 
    //     std::ostream_iterator<int>(std::cout, " ")
    // );
    printf("\nNumber of triangles : %d\n", tris.size());
    printf("Number of poses : %d\n", num_images);

    size_t real_width = width;
    size_t real_height = height;

    // atomic min only support int32
    
    // Create device outputs
    thrust::device_vector<int> device_pose_occluded(num_images, 0);
    thrust::device_vector<int> device_pose_occluded_other(num_images, 0);
    thrust::device_vector<float> device_pose_clutter_points(num_images, 0);
    thrust::device_vector<float> device_pose_total_points(num_images, 0);

    device_vector_holder<int32_t> device_depth_int(num_images*real_width*real_height, INT_MAX);
    // thrust::device_vector<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);
    thrust::device_vector<int32_t> device_lock_int(num_images*real_width*real_height, 0);
    thrust::device_vector<uint8_t> device_red_int(num_images*real_width*real_height, 0);
    thrust::device_vector<uint8_t> device_green_int(num_images*real_width*real_height, 0);
    thrust::device_vector<uint8_t> device_blue_int(num_images*real_width*real_height, 0);

  
    Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
    Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());

    // Mapping each pose to model
    int* device_pose_model_map_ptr = thrust::raw_pointer_cast(device_pose_model_map.data());

    // Mapping each model to triangle range
    int* device_tris_model_count_low_ptr = thrust::raw_pointer_cast(device_tris_model_count_low.data());
    int* device_tris_model_count_high_ptr = thrust::raw_pointer_cast(device_tris_model_count_high.data());
    // int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());

    int* device_pose_occluded_vec = thrust::raw_pointer_cast(device_pose_occluded.data());
    int* device_pose_occluded_other_vec = thrust::raw_pointer_cast(device_pose_occluded_other.data());
    float* device_pose_clutter_points_vec = thrust::raw_pointer_cast(device_pose_clutter_points.data());
    float* device_pose_total_points_vec = thrust::raw_pointer_cast(device_pose_total_points.data());

    int32_t* device_source_depth_vec = thrust::raw_pointer_cast(device_source_depth.data());
    uint8_t* device_source_red_vec = thrust::raw_pointer_cast(device_source_color_red.data());
    uint8_t* device_source_green_vec = thrust::raw_pointer_cast(device_source_color_green.data());
    uint8_t* device_source_blue_vec = thrust::raw_pointer_cast(device_source_color_blue.data());

    int32_t* depth_image_vec = device_depth_int.data();
    int32_t* lock_int_vec = thrust::raw_pointer_cast(device_lock_int.data());
    uint8_t* red_image_vec = thrust::raw_pointer_cast(device_red_int.data());
    uint8_t* green_image_vec = thrust::raw_pointer_cast(device_green_int.data());
    uint8_t* blue_image_vec = thrust::raw_pointer_cast(device_blue_int.data());

    // Initialize rendered images with source images
    dim3 block(16,16);
    dim3 grid((real_width*num_images + block.x - 1)/block.x, (real_height + block.y - 1)/block.y);
    // copy_source_to_render<<<grid,block>>>(red_image_vec,green_image_vec,blue_image_vec,
    //                             depth_image_vec,
    //                             device_source_red_vec, device_source_green_vec, device_source_blue_vec,
    //                             device_source_depth_vec,
    //                             width,height,num_images);
    // hipDeviceSynchronize();

    // Render all poses
    dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
    render_triangle_multi<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                    device_poses_ptr, poses.size(),
                                                    depth_image_vec, width, height, 
                                                    device_pose_model_map_ptr, device_tris_model_count_low_ptr,
                                                    device_tris_model_count_high_ptr,
                                                    proj_mat, roi,
                                                    red_image_vec,green_image_vec,blue_image_vec,
                                                    device_source_depth_vec,
                                                    device_source_red_vec, device_source_green_vec, device_source_blue_vec,
                                                    device_pose_occluded_vec,
                                                    device_single_result_image,
                                                    lock_int_vec,
                                                    device_pose_occluded_other_vec,
                                                    device_pose_clutter_points_vec,
                                                    device_pose_total_points_vec);
    // hipDeviceSynchronize();
    // Objects occluding other objects already in the scene
    printf("Pose Occlusions\n");
    thrust::copy(
        device_pose_occluded.begin(),
        device_pose_occluded.end(), 
        std::ostream_iterator<int>(std::cout, " ")
    );
    printf("\n");
    thrust::copy(device_pose_occluded.begin(), device_pose_occluded.end(), pose_occluded.begin());
    
    // Objects occluded by existing objects in the scene, need to do ICP again for these
    printf("Pose Occlusions Other\n");
    thrust::copy(
        device_pose_occluded_other.begin(),
        device_pose_occluded_other.end(), 
        std::ostream_iterator<int>(std::cout, " ")
    );
    printf("\n");
    thrust::copy(device_pose_occluded_other.begin(), device_pose_occluded_other.end(), pose_occluded_other.begin());

    printf("Pose Clutter Ratio\n");
    thrust::transform(
        device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
        device_pose_total_points.begin(), device_pose_clutter_points.begin(), 
        thrust::divides<float>()
    );
    thrust::device_vector<float> rendered_multiplier_val(num_images, 100);
    thrust::transform(
        device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
        rendered_multiplier_val.begin(), device_pose_clutter_points.begin(), 
        thrust::multiplies<float>()
    );
    thrust::copy(device_pose_clutter_points.begin(), device_pose_clutter_points.end(), clutter_cost.begin());

    thrust::copy(
        device_pose_clutter_points.begin(),
        device_pose_clutter_points.end(), 
        std::ostream_iterator<float>(std::cout, " ")
    );
    printf("\n");

    result_depth.resize(num_images*real_width*real_height);
    {
        thrust::device_vector<int32_t> v3(depth_image_vec, depth_image_vec + num_images*real_width*real_height);
        thrust::transform(v3.begin(), v3.end(),v3.begin(), max2zero_functor());
        thrust::copy(v3.begin(), v3.end(), result_depth.begin());

    }
    
    std::vector<uint8_t> result_red(num_images*real_width*real_height);
    std::vector<uint8_t> result_green(num_images*real_width*real_height);
    std::vector<uint8_t> result_blue(num_images*real_width*real_height);
    {
        thrust::transform(device_red_int.begin(), device_red_int.end(),
                          device_red_int.begin(), max2zero_functor());
        thrust::copy(device_red_int.begin(), device_red_int.end(), result_red.begin());
        thrust::transform(device_green_int.begin(), device_green_int.end(),
                          device_green_int.begin(), max2zero_functor());
        thrust::copy(device_green_int.begin(), device_green_int.end(), result_green.begin());
        thrust::transform(device_blue_int.begin(), device_blue_int.end(),
                          device_blue_int.begin(), max2zero_functor());
        thrust::copy(device_blue_int.begin(), device_blue_int.end(), result_blue.begin());

    }
    if (result_color.size() > 0) result_color.clear();
    result_color.push_back(result_red);
    result_color.push_back(result_green);
    result_color.push_back(result_blue);


    thrust::transform(device_depth_int.begin_thr(), device_depth_int.end_thr(),
                      device_depth_int.begin_thr(), max2zero_functor());
    return device_depth_int;
}

device_vector_holder<int> render_cuda(const std::vector<Model::Triangle>& tris,const std::vector<Model::mat4x4>& poses,
                            size_t width, size_t height, const Model::mat4x4& proj_mat,
                            std::vector<int32_t>& result_depth, std::vector<std::vector<uint8_t>>& result_color, const Model::ROI roi){

    const size_t threadsPerBlock = 256;
    // std::cout <<tris[0].color.v1;
    thrust::device_vector<Model::Triangle> device_tris = tris;
    thrust::device_vector<Model::mat4x4> device_poses = poses;

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
        assert(roi.x + roi.width <= width && "roi out of image");
        assert(roi.y + roi.height <= height && "roi out of image");
    }
    // atomic min only support int32
    
    device_vector_holder<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);

    // thrust::device_vector<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);
    thrust::device_vector<uint8_t> device_red_int(poses.size()*real_width*real_height, 0);
    thrust::device_vector<uint8_t> device_green_int(poses.size()*real_width*real_height, 0);
    thrust::device_vector<uint8_t> device_blue_int(poses.size()*real_width*real_height, 0);
    // thrust::device_vector<float> l(poses.size()*real_width*real_height, 0);
    // thrust::device_vector<float> a(poses.size()*real_width*real_height, 0);
    // thrust::device_vector<float> b(poses.size()*real_width*real_height, 0);
    // {
        Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
        Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());
        // int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());
        int32_t* depth_image_vec = device_depth_int.data();
        uint8_t* red_image_vec = thrust::raw_pointer_cast(device_red_int.data());
        uint8_t* green_image_vec = thrust::raw_pointer_cast(device_green_int.data());
        uint8_t* blue_image_vec = thrust::raw_pointer_cast(device_blue_int.data());
        // float* l_vec = thrust::raw_pointer_cast(l.data());
        // float* a_vec = thrust::raw_pointer_cast(a.data());
        // float* b_vec = thrust::raw_pointer_cast(b.data());

        dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
        render_triangle<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                        device_poses_ptr, poses.size(),
                                                        depth_image_vec, width, height, proj_mat, roi,
                                                        red_image_vec,green_image_vec,blue_image_vec);
        hipDeviceSynchronize();
        // gpuErrchk(hipPeekAtLastError());
    // }


    result_depth.resize(poses.size()*real_width*real_height);
    {
        thrust::device_vector<int32_t> v3(depth_image_vec, depth_image_vec + poses.size()*real_width*real_height);
        thrust::transform(v3.begin(), v3.end(),v3.begin(), max2zero_functor());
        thrust::copy(v3.begin(), v3.end(), result_depth.begin());

    }
    
    std::vector<uint8_t> result_red(poses.size()*real_width*real_height);
    std::vector<uint8_t> result_green(poses.size()*real_width*real_height);
    std::vector<uint8_t> result_blue(poses.size()*real_width*real_height);
    {
        thrust::transform(device_red_int.begin(), device_red_int.end(),
                          device_red_int.begin(), max2zero_functor());
        thrust::copy(device_red_int.begin(), device_red_int.end(), result_red.begin());
        thrust::transform(device_green_int.begin(), device_green_int.end(),
                          device_green_int.begin(), max2zero_functor());
        thrust::copy(device_green_int.begin(), device_green_int.end(), result_green.begin());
        thrust::transform(device_blue_int.begin(), device_blue_int.end(),
                          device_blue_int.begin(), max2zero_functor());
        thrust::copy(device_blue_int.begin(), device_blue_int.end(), result_blue.begin());

    }
    result_color.push_back(result_red);
    result_color.push_back(result_green);
    result_color.push_back(result_blue);

    // device_vector_holder<int> device_depth_int_v(poses.size()*real_width*real_height, result_depth);
    // device_vector_holder<int> device_depth_int_v = device_depth_int;
    // device_vector_holder<int> device_depth_int_v;
    // thrust::copy(device_depth_int.begin(), device_depth_int.end(), device_depth_int_v.begin());
    
    // device_depth_int_v.__gpu_memory = thrust::raw_pointer_cast(device_depth_int.data());
    // device_depth_int_v.__size = device_depth_int.size();
    // device_depth_int_v.valid = true;
    // return device_depth_int_v;
    // thrust::transform(device_depth_int.begin(), device_depth_int.end(),device_depth_int.begin(), max2zero_functor());
    thrust::transform(device_depth_int.begin_thr(), device_depth_int.end_thr(),
                      device_depth_int.begin_thr(), max2zero_functor());
    return device_depth_int;
}

}

