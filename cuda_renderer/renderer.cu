#include "hip/hip_runtime.h"
#include "cuda_renderer/common.cuh"
#include "cuda_renderer/image_renderer.cuh"
#include "cuda_renderer/compute_point_clouds.cuh"
#include "cuda_renderer/compute_costs.cuh"
#include "cuda_renderer/renderer.h"
#include "cuda_renderer/utils.cuh"
#include <fast_gicp/cuda/brute_force_knn.cuh>
#include <fast_gicp/cuda/fast_gicp_cuda.cuh>

// #include <math.h> 
#include "hip/hip_fp16.h"
// #include <numeric> 
#define SQR(x) ((x)*(x))
#define POW2(x) SQR(x)
#define POW3(x) ((x)*(x)*(x))
#define POW4(x) (POW2(x)*POW2(x))
#define POW7(x) (POW3(x)*POW3(x)*(x))
#define DegToRad(x) ((x)*M_PI/180)
#define RadToDeg(x) ((x)/M_PI*180)
#define USE_TREE 0
#define USE_CLUTTER 0

namespace cuda_renderer {
    static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
    {
        if(err!=hipSuccess)
        {
            fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
            std::cin.get();
            exit(EXIT_FAILURE);
        }
    }
    #define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)


    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
    {
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
    }


    template<typename T>
    device_vector_holder<T>::~device_vector_holder(){
        __free();
    }

    template<typename T>
    void device_vector_holder<T>::__free(){
        if(valid){
            hipFree(__gpu_memory);
            valid = false;
            __size = 0;
        }
    }

    template<typename T>
    device_vector_holder<T>::device_vector_holder(size_t size_, T init)
    {
        __malloc(size_);
        thrust::fill(begin_thr(), end_thr(), init);
    }

    template<typename T>
    void device_vector_holder<T>::__malloc(size_t size_){
        if(valid) __free();
        hipMalloc((void**)&__gpu_memory, size_ * sizeof(T));
        __size = size_;
        valid = true;
    }

    template<typename T>
    device_vector_holder<T>::device_vector_holder(size_t size_){
        __malloc(size_);
    }

    template class device_vector_holder<int>;

    

    struct max2zero_functor{

        max2zero_functor(){}

        __host__ __device__
        int32_t operator()(const int32_t& x) const
        {
            return (x==INT_MAX)? 0: x;
        }
    };

    struct cost_percentage_functor{

        cost_percentage_functor(){}

        __host__ __device__
        float operator()(const float& x, const float& y) const
        {
            if (y == 0)
            {
                return -1;
            }
            else
            {
                return x/y;
            }
        }
    };

    struct cost_multiplier_functor{

        cost_multiplier_functor(){}

        __host__ __device__
        float operator()(const float& x, const float& y) const
        {
            if (x == -1)
            {
                return -1;
            }
            else
            {
                return x*y;
            }
        }
    };

    __device__
    void rasterization_with_source(const Model::Triangle dev_tri, Model::float3 last_row,
                                            int32_t* depth_entry, size_t width, size_t height,
                                            const Model::ROI roi, 
                                            uint8_t* red_entry,uint8_t* green_entry,uint8_t* blue_entry,
                                            int32_t* source_depth_entry,
                                            uint8_t* source_red_entry,uint8_t* source_green_entry,uint8_t* source_blue_entry,
                                            int* pose_occluded_entry,
                                            int32_t* lock_entry,
                                            int* pose_occluded_other_entry,
                                            float* pose_clutter_points_entry,
                                            float* pose_total_points_entry,
                                            uint8_t* source_label_entry,
                                            int* pose_segmentation_label_entry,
                                            bool use_segmentation_label,
                                            float occlusion_threshold) {
                                            // float* l_entry,float* a_entry,float* b_entry){
        // refer to tiny renderer
        // https://github.com/ssloy/tinyrenderer/blob/master/our_gl.cpp
        float pts2[3][2];

        // viewport transform(0, 0, width, height)
        pts2[0][0] = dev_tri.v0.x/last_row.x*width/2.0f+width/2.0f; pts2[0][1] = dev_tri.v0.y/last_row.x*height/2.0f+height/2.0f;
        pts2[1][0] = dev_tri.v1.x/last_row.y*width/2.0f+width/2.0f; pts2[1][1] = dev_tri.v1.y/last_row.y*height/2.0f+height/2.0f;
        pts2[2][0] = dev_tri.v2.x/last_row.z*width/2.0f+width/2.0f; pts2[2][1] = dev_tri.v2.y/last_row.z*height/2.0f+height/2.0f;

        float bboxmin[2] = {FLT_MAX,  FLT_MAX};
        float bboxmax[2] = {-FLT_MAX, -FLT_MAX};

        float clamp_max[2] = {float(width-1), float(height-1)};
        float clamp_min[2] = {0, 0};

        size_t real_width = width;
        if(roi.width > 0 && roi.height > 0){  // depth will be flipped
            clamp_min[0] = roi.x;
            clamp_min[1] = height-1 - (roi.y + roi.height - 1);
            clamp_max[0] = (roi.x + roi.width) - 1;
            clamp_max[1] = height-1 - roi.y;
            real_width = roi.width;
        }


        for (int i=0; i<3; i++) {
            for (int j=0; j<2; j++) {
                bboxmin[j] = std__max(clamp_min[j], std__min(bboxmin[j], pts2[i][j]));
                bboxmax[j] = std__min(clamp_max[j], std__max(bboxmax[j], pts2[i][j]));
            }
        }

        size_t P[2];
        for(P[1] = size_t(bboxmin[1]+0.5f); P[1]<=bboxmax[1]; P[1] += 1){
            for(P[0] = size_t(bboxmin[0]+0.5f); P[0]<=bboxmax[0]; P[0] += 1){
                Model::float3 bc_screen  = barycentric(pts2[0], pts2[1], pts2[2], P);

                if (bc_screen.x<-0.0f || bc_screen.y<-0.0f || bc_screen.z<-0.0f ||
                        bc_screen.x>1.0f || bc_screen.y>1.0f || bc_screen.z>1.0f ) continue;

                Model::float3 bc_over_z = {bc_screen.x/last_row.x, bc_screen.y/last_row.y, bc_screen.z/last_row.z};

                // refer to https://en.wikibooks.org/wiki/Cg_Programming/Rasterization, Perspectively Correct Interpolation
    //            float frag_depth = (dev_tri.v0.z * bc_over_z.x + dev_tri.v1.z * bc_over_z.y + dev_tri.v2.z * bc_over_z.z)
    //                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

                // this seems better
                float frag_depth = (bc_screen.x + bc_screen.y + bc_screen.z)
                        /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

                size_t x_to_write = (P[0] + roi.x);
                size_t y_to_write = (height-1 - P[1] - roi.y);
                int32_t curr_depth = int32_t(frag_depth/**1000*/ + 0.5f);
                // printf("x:%d, y:%d, depth:%d\n", x_to_write, y_to_write, curr_depth);
                int32_t& depth_to_write = depth_entry[x_to_write+y_to_write*real_width];
                int32_t& source_depth = source_depth_entry[x_to_write+y_to_write*real_width];
                uint8_t source_red = source_red_entry[x_to_write+y_to_write*real_width];
                uint8_t source_green = source_green_entry[x_to_write+y_to_write*real_width];
                uint8_t source_blue = source_blue_entry[x_to_write+y_to_write*real_width];
                uint8_t source_label = source_label_entry[x_to_write+y_to_write*real_width];

                // if(depth_to_write > curr_depth){
                //     red_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v0);
                //     green_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v1);
                //     blue_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v2);
                // }
                // atomicMin(&depth_to_write, curr_depth);
                bool wait = true;
                while(wait){
                    if(0 == atomicExch(&lock_entry[x_to_write+y_to_write*real_width], 1)){
                        if(curr_depth < depth_entry[x_to_write+y_to_write*real_width]){
                            // occluding an existing point of same object
                            depth_entry[x_to_write+y_to_write*real_width] = curr_depth;
                            red_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v0);
                            green_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v1);
                            blue_entry[x_to_write+y_to_write*real_width] = (uint8_t)(dev_tri.color.v2);
                        }
                        lock_entry[x_to_write+y_to_write*real_width] = 0;
                        wait = false;
                    }
                }
                // 1.0 is 1cm occlusion threshold
                int32_t& new_depth = depth_entry[x_to_write+y_to_write*real_width];
                if ((use_segmentation_label == false && abs(new_depth - source_depth) > occlusion_threshold) ||
                    (use_segmentation_label == true && 
                     *pose_segmentation_label_entry != source_label && abs(new_depth - source_depth) > 0.5))
                {
                    // printf("%d, %d\n", *pose_segmentation_label_entry, source_label);
                    // printf("%d, %d\n", source_depth, curr_depth);
                    if(new_depth > source_depth && source_depth > 0){
                        // when we are rendering at x,y where source pixel is also present at depth closer to camera
                        // valid condition as source occludes render
                        // if (false)
                        // {
                        //     // add source pixels
                        //     red_entry[x_to_write+y_to_write*real_width] = source_red;
                        //     green_entry[x_to_write+y_to_write*real_width] = source_green;
                        //     blue_entry[x_to_write+y_to_write*real_width] = source_blue;
                        //     atomicMin(&new_depth, source_depth);
                        // }
                        // else
                        // {
                            // add black
                            red_entry[x_to_write+y_to_write*real_width] = 0;
                            green_entry[x_to_write+y_to_write*real_width] = 0;
                            blue_entry[x_to_write+y_to_write*real_width] = 0;
                            atomicMax(&new_depth, INT_MAX);
                            if (USE_TREE)
                                atomicOr(pose_occluded_other_entry, 1);
                            if (USE_CLUTTER)
                            {
                                if (source_depth <=  new_depth - 5)
                                {
                                    atomicAdd(pose_clutter_points_entry, 1);
                                }
                            }
                    }
                    // invalid condition where source pixel is behind and we are rendering a pixel at same x,y with lesser depth 
                    else if(new_depth <= source_depth && source_depth > 0){
                        // invalid as render occludes source
                        if (USE_TREE)
                            atomicOr(pose_occluded_entry, 1);
                        // printf("Occlusion\n");
                    }
                }
                if (USE_CLUTTER)
                    atomicAdd(pose_total_points_entry, 1);

            }
        }
    }


    __global__ void render_triangle_multi(
                                    Model::Triangle* device_tris_ptr, size_t device_tris_size,
                                    Model::mat4x4* device_poses_ptr, size_t device_poses_size,
                                    int32_t* depth_image_vec, size_t width, size_t height,
                                    int* device_pose_model_map_ptr, int* device_tris_model_count_low_ptr,  
                                    int* device_tris_model_count_high_ptr,
                                    const Model::mat4x4 proj_mat, const Model::ROI roi,
                                    uint8_t* red_image_vec,uint8_t* green_image_vec,uint8_t* blue_image_vec,
                                    int32_t* device_source_depth_vec,
                                    uint8_t* device_source_red_vec,uint8_t* device_source_green_vec,uint8_t* device_source_blue_vec,
                                    int* pose_occluded_vec,
                                    int* device_single_result_image,
                                    int32_t* lock_int_vec,
                                    int* pose_occluded_other_vec,
                                    float* pose_clutter_points_vec, 
                                    float* pose_total_points_vec,
                                    uint8_t* device_source_mask_label_vec,
                                    int* pose_segmentation_label_vec,
                                    bool use_segmentation_label,
                                    float occlusion_threshold) {
        size_t pose_i = blockIdx.y;
        int model_id = device_pose_model_map_ptr[pose_i];
        size_t tri_i = blockIdx.x*blockDim.x + threadIdx.x;

        if(tri_i>=device_tris_size) return;

        if (!(tri_i < device_tris_model_count_high_ptr[model_id] && tri_i >= device_tris_model_count_low_ptr[model_id]))
            return; 

        size_t real_width = width;
        size_t real_height = height;
        if(roi.width > 0 && roi.height > 0){
            real_width = roi.width;
            real_height = roi.height;
        }
        int32_t* depth_entry;
        int32_t* lock_entry;
        uint8_t* red_entry;
        uint8_t* green_entry;
        uint8_t* blue_entry;
        int* pose_occluded_entry;
        int* pose_occluded_other_entry;
        float* pose_clutter_points_entry;
        float* pose_total_points_entry;
        int* pose_segmentation_label_entry = NULL;
        // printf("device_single_result_image:%d\n",device_single_result_image);
        if (*device_single_result_image)
        {
            depth_entry = depth_image_vec; //length: width*height 32bits int
            red_entry = red_image_vec;
            green_entry = green_image_vec;
            blue_entry = blue_image_vec;
            pose_occluded_entry = pose_occluded_vec;
            lock_entry = lock_int_vec;
            pose_occluded_other_entry = pose_occluded_other_vec;
            pose_clutter_points_entry = pose_clutter_points_vec;
            pose_total_points_entry = pose_total_points_vec;
            if (pose_segmentation_label_vec != NULL)
                pose_segmentation_label_entry = pose_segmentation_label_vec;
        }
        else
        {
            depth_entry = depth_image_vec + pose_i*real_width*real_height; //length: width*height 32bits int
            lock_entry = lock_int_vec + pose_i*real_width*real_height;
            red_entry = red_image_vec + pose_i*real_width*real_height;
            green_entry = green_image_vec + pose_i*real_width*real_height;
            blue_entry = blue_image_vec + pose_i*real_width*real_height;
            pose_occluded_entry = pose_occluded_vec + pose_i;
            pose_occluded_other_entry = pose_occluded_other_vec + pose_i;
            pose_clutter_points_entry = pose_clutter_points_vec + pose_i;
            pose_total_points_entry = pose_total_points_vec + pose_i;
            if (pose_segmentation_label_vec != NULL)
                pose_segmentation_label_entry = pose_segmentation_label_vec + pose_i;
        }
        

        Model::mat4x4* pose_entry = device_poses_ptr + pose_i; // length: 16 32bits float
        Model::Triangle* tri_entry = device_tris_ptr + tri_i; // length: 9 32bits float

        // model transform
        Model::Triangle local_tri = transform_triangle(*tri_entry, *pose_entry);

        // assume last column of projection matrix is  0 0 1 0
        Model::float3 last_row = {
            local_tri.v0.z,
            local_tri.v1.z,
            local_tri.v2.z
        };
        // projection transform
        local_tri = transform_triangle(local_tri, proj_mat);

        // rasterization(local_tri, last_row, depth_entry, width, height, roi,red_entry,green_entry,blue_entry);
        rasterization_with_source(
            local_tri, last_row, depth_entry, width, height, roi,
            red_entry,green_entry,blue_entry,
            device_source_depth_vec,
            device_source_red_vec, device_source_green_vec, device_source_blue_vec,
            pose_occluded_entry,
            lock_entry,
            pose_occluded_other_entry,
            pose_clutter_points_entry,
            pose_total_points_entry,
            device_source_mask_label_vec,
            pose_segmentation_label_entry,
            use_segmentation_label,
            occlusion_threshold);
    }



    device_vector_holder<int> render_cuda_multi(
                                const std::vector<Model::Triangle>& tris,
                                const std::vector<Model::mat4x4>& poses,
                                const std::vector<int> pose_model_map,
                                const std::vector<int> tris_model_count,
                                size_t width, size_t height, const Model::mat4x4& proj_mat,
                                const std::vector<int32_t>& source_depth,
                                const std::vector<std::vector<uint8_t>>& source_color,
                                std::vector<int32_t>& result_depth, 
                                std::vector<std::vector<uint8_t>>& result_color,
                                std::vector<int>& pose_occluded,
                                int single_result_image,
                                std::vector<int>& pose_occluded_other,
                                std::vector<float>& clutter_cost,
                                const std::vector<uint8_t>& source_mask_label,
                                const std::vector<int>& pose_segmentation_label) {

        // Create device inputs
        int* device_single_result_image;
        hipMalloc((void**)&device_single_result_image, sizeof(int));
        hipMemcpy(device_single_result_image, &single_result_image, sizeof(int), hipMemcpyHostToDevice);
        int num_images;
        if (single_result_image)
        {
            num_images = 1;
        }
        else
        {
            num_images = poses.size();
        }
        const Model::ROI roi= {0, 0, 0, 0};
        const size_t threadsPerBlock = 256;
        // std::cout <<tris[0].color.v1;
        thrust::device_vector<Model::Triangle> device_tris = tris;
        thrust::device_vector<Model::mat4x4> device_poses = poses;
        thrust::device_vector<int> device_tris_model_count_low = tris_model_count;
        thrust::device_vector<int> device_tris_model_count_high = tris_model_count;
        thrust::device_vector<int> device_pose_model_map = pose_model_map;
        thrust::device_vector<int> device_pose_segmentation_label = pose_segmentation_label;

        thrust::device_vector<int32_t> device_source_depth = source_depth;
        thrust::device_vector<uint8_t> device_source_color_red = source_color[0];
        thrust::device_vector<uint8_t> device_source_color_green = source_color[1];
        thrust::device_vector<uint8_t> device_source_color_blue = source_color[2];
        thrust::device_vector<uint8_t> device_source_mask_label = source_mask_label;

        // thrust::copy(
        //     device_tris_model_count.begin(),
        //     device_tris_model_count.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        printf("\nPose segmentation label : \n");
        thrust::copy(
            device_pose_segmentation_label.begin(),
            device_pose_segmentation_label.end(), 
            std::ostream_iterator<int>(std::cout, " ")
        );
        thrust::exclusive_scan(
            device_tris_model_count_low.begin(), device_tris_model_count_low.end(), 
            device_tris_model_count_low.begin(), 0
        ); // in-place scan
        thrust::inclusive_scan(
            device_tris_model_count_high.begin(), device_tris_model_count_high.end(), 
            device_tris_model_count_high.begin()
        ); // in-place scan
        // thrust::copy(
        //     device_tris_model_count_low.begin(),
        //     device_tris_model_count_low.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\n");
        // thrust::copy(
        //     device_tris_model_count_high.begin(),
        //     device_tris_model_count_high.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\n");
        // thrust::copy(
        //     device_pose_model_map.begin(),
        //     device_pose_model_map.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        printf("\nNumber of triangles : %d\n", tris.size());
        printf("Number of poses : %d\n", num_images);

        size_t real_width = width;
        size_t real_height = height;

        // atomic min only support int32
        
        // Create device outputs
        thrust::device_vector<int> device_pose_occluded(num_images, 0);
        thrust::device_vector<int> device_pose_occluded_other(num_images, 0);
        thrust::device_vector<float> device_pose_clutter_points(num_images, 0);
        thrust::device_vector<float> device_pose_total_points(num_images, 0);

        device_vector_holder<int32_t> device_depth_int(num_images*real_width*real_height, INT_MAX);
        // thrust::device_vector<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);
        thrust::device_vector<int32_t> device_lock_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_red_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_green_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_blue_int(num_images*real_width*real_height, 0);

    
        Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
        Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());

        // Mapping each pose to model
        int* device_pose_model_map_ptr = thrust::raw_pointer_cast(device_pose_model_map.data());

        // Mapping each model to triangle range
        int* device_tris_model_count_low_ptr = thrust::raw_pointer_cast(device_tris_model_count_low.data());
        int* device_tris_model_count_high_ptr = thrust::raw_pointer_cast(device_tris_model_count_high.data());
        // int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());

        int* device_pose_occluded_vec = thrust::raw_pointer_cast(device_pose_occluded.data());
        int* device_pose_occluded_other_vec = thrust::raw_pointer_cast(device_pose_occluded_other.data());
        float* device_pose_clutter_points_vec = thrust::raw_pointer_cast(device_pose_clutter_points.data());
        float* device_pose_total_points_vec = thrust::raw_pointer_cast(device_pose_total_points.data());
        int* device_pose_segmentation_label_vec = thrust::raw_pointer_cast(device_pose_segmentation_label.data());
        bool use_segmentation_label = false;
        if (device_pose_segmentation_label.size() > 0)
            use_segmentation_label = true ;

        int32_t* device_source_depth_vec = thrust::raw_pointer_cast(device_source_depth.data());
        uint8_t* device_source_red_vec = thrust::raw_pointer_cast(device_source_color_red.data());
        uint8_t* device_source_green_vec = thrust::raw_pointer_cast(device_source_color_green.data());
        uint8_t* device_source_blue_vec = thrust::raw_pointer_cast(device_source_color_blue.data());
        uint8_t* device_source_mask_label_vec = thrust::raw_pointer_cast(device_source_mask_label.data());

        int32_t* depth_image_vec = device_depth_int.data();
        int32_t* lock_int_vec = thrust::raw_pointer_cast(device_lock_int.data());
        uint8_t* red_image_vec = thrust::raw_pointer_cast(device_red_int.data());
        uint8_t* green_image_vec = thrust::raw_pointer_cast(device_green_int.data());
        uint8_t* blue_image_vec = thrust::raw_pointer_cast(device_blue_int.data());

        // Initialize rendered images with source images
        dim3 block(16,16);
        dim3 grid((real_width*num_images + block.x - 1)/block.x, (real_height + block.y - 1)/block.y);
        // copy_source_to_render<<<grid,block>>>(red_image_vec,green_image_vec,blue_image_vec,
        //                             depth_image_vec,
        //                             device_source_red_vec, device_source_green_vec, device_source_blue_vec,
        //                             device_source_depth_vec,
        //                             width,height,num_images);
        // hipDeviceSynchronize();

        // Render all poses
        dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
        render_triangle_multi<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                        device_poses_ptr, poses.size(),
                                                        depth_image_vec, width, height, 
                                                        device_pose_model_map_ptr, device_tris_model_count_low_ptr,
                                                        device_tris_model_count_high_ptr,
                                                        proj_mat, roi,
                                                        red_image_vec,green_image_vec,blue_image_vec,
                                                        device_source_depth_vec,
                                                        device_source_red_vec, device_source_green_vec, device_source_blue_vec,
                                                        device_pose_occluded_vec,
                                                        device_single_result_image,
                                                        lock_int_vec,
                                                        device_pose_occluded_other_vec,
                                                        device_pose_clutter_points_vec,
                                                        device_pose_total_points_vec,
                                                        device_source_mask_label_vec,
                                                        device_pose_segmentation_label_vec,
                                                        use_segmentation_label,
                                                        1.0);
        // hipDeviceSynchronize();
        // Objects occluding other objects already in the scene
        if (USE_TREE)
        {
            printf("Pose Occlusions\n");
            thrust::copy(
                device_pose_occluded.begin(),
                device_pose_occluded.end(), 
                std::ostream_iterator<int>(std::cout, " ")
            );
            printf("\n");
            thrust::copy(device_pose_occluded.begin(), device_pose_occluded.end(), pose_occluded.begin());
            
            // Objects occluded by existing objects in the scene, need to do ICP again for these
            printf("Pose Occlusions Other\n");
            thrust::copy(
                device_pose_occluded_other.begin(),
                device_pose_occluded_other.end(), 
                std::ostream_iterator<int>(std::cout, " ")
            );
            printf("\n");
            thrust::copy(device_pose_occluded_other.begin(), device_pose_occluded_other.end(), pose_occluded_other.begin());
        }
        if (USE_CLUTTER)
        {
            // printf("Pose Clutter Ratio\n");
            thrust::transform(
                device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
                device_pose_total_points.begin(), device_pose_clutter_points.begin(), 
                thrust::divides<float>()
            );
            thrust::device_vector<float> rendered_multiplier_val(num_images, 100);
            thrust::transform(
                device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
                rendered_multiplier_val.begin(), device_pose_clutter_points.begin(), 
                thrust::multiplies<float>()
            );
            thrust::copy(device_pose_clutter_points.begin(), device_pose_clutter_points.end(), clutter_cost.begin());
        }
        // thrust::copy(
        //     device_pose_clutter_points.begin(),
        //     device_pose_clutter_points.end(), 
        //     std::ostream_iterator<float>(std::cout, " ")
        // );
        printf("\n");

        result_depth.resize(num_images*real_width*real_height);
        {
            thrust::device_vector<int32_t> v3(depth_image_vec, depth_image_vec + num_images*real_width*real_height);
            thrust::transform(v3.begin(), v3.end(),v3.begin(), max2zero_functor());
            thrust::copy(v3.begin(), v3.end(), result_depth.begin());

        }
        
        std::vector<uint8_t> result_red(num_images*real_width*real_height);
        std::vector<uint8_t> result_green(num_images*real_width*real_height);
        std::vector<uint8_t> result_blue(num_images*real_width*real_height);
        {
            thrust::transform(device_red_int.begin(), device_red_int.end(),
                            device_red_int.begin(), max2zero_functor());
            thrust::copy(device_red_int.begin(), device_red_int.end(), result_red.begin());
            thrust::transform(device_green_int.begin(), device_green_int.end(),
                            device_green_int.begin(), max2zero_functor());
            thrust::copy(device_green_int.begin(), device_green_int.end(), result_green.begin());
            thrust::transform(device_blue_int.begin(), device_blue_int.end(),
                            device_blue_int.begin(), max2zero_functor());
            thrust::copy(device_blue_int.begin(), device_blue_int.end(), result_blue.begin());

        }
        if (result_color.size() > 0) result_color.clear();
        result_color.push_back(result_red);
        result_color.push_back(result_green);
        result_color.push_back(result_blue);


        thrust::transform(device_depth_int.begin_thr(), device_depth_int.end_thr(),
                        device_depth_int.begin_thr(), max2zero_functor());
        return device_depth_int;
    }

    // __global__ void compute_observed_cost_cylinder(
    //     int num_poses,
    //     int observed_cloud_point_num,
    //     uint8_t* cuda_observed_explained,
    //     float* cuda_observed_cloud,
    //     size_t cloud_pitch,
    //     Model::mat4x4* device_poses_ptr,
    //     float* pose_cylinder_radius,
    //     float* observed_total_unexplained)
    // {
    //     /*
    //      * @observed_cloud_point_num - number of points in observed scene
    //      * @cuda_observed_explained (num_poses x observed_point_num) 
    //      *      - binary value indicating whether given point is explained or not based on distance
    //      */
    //     size_t point_index = blockIdx.x*blockDim.x + threadIdx.x;
    //     if(point_index >= num_poses * observed_cloud_point_num) return;

    //     size_t pose_index = point_index/observed_cloud_point_num;
    //     size_t cloud_point_index = point_index % observed_cloud_point_num;

    //     float cylinder_radius = pose_cylinder_radius[pose_index];
    //     Model::mat4x4 pose_entry = device_poses_ptr[pose_index];

    //     float pose_x = pose_entry.a3;
    //     float pose_y = pose_entry.b3;
    //     float pose_z = pose_entry.c3;

    //     float* row_0 = (float *)((char*)cuda_observed_cloud + 0 * cloud_pitch);
    //     float* row_1 = (float *)((char*)cuda_observed_cloud + 1 * cloud_pitch);
    //     float* row_2 = (float *)((char*)cuda_observed_cloud + 2 * cloud_pitch);
    //     float point_x = row_0[cloud_point_index];
    //     float point_y = row_1[cloud_point_index];
    //     float point_z = row_2[cloud_point_index];

    //     // printf("pose_x:%f, pose_y:%f, point_x:%f, point_y:%f, distance:\n", pose_x, pose_y, point_x, point_y);

    //     if (sqrt((point_x - pose_x)*(point_x - pose_x) + (point_y - pose_y)*(point_y - pose_y)) <  cylinder_radius)
    //     {
    //         // lies within the inscribed volume of pose
    //         if (cuda_observed_explained[point_index] == 0)
    //         {
    //             atomicAdd(&observed_total_unexplained[pose_index], 1);
    //         }
    //     }


    // }
    void render_cuda_multi_unified_old(
        const std::string stage,
        const std::vector<Model::Triangle>& tris,
        const std::vector<Model::mat4x4>& poses,
        const std::vector<int> pose_model_map,
        const std::vector<int> tris_model_count,
        size_t width, size_t height, const Model::mat4x4& proj_mat,
        const std::vector<int32_t>& source_depth,
        const std::vector<std::vector<uint8_t>>& source_color,
        int single_result_image,
        std::vector<float>& clutter_cost,
        const std::vector<uint8_t>& source_mask_label,
        const std::vector<int>& pose_segmentation_label,
        int stride,
        int point_dim,
        int depth_factor,
        float kCameraCX,
        float kCameraCY,
        float kCameraFX,
        float kCameraFY,
        float* observed_depth,
        uint8_t* observed_color,
        int observed_point_num,
        std::vector<float> pose_observed_points_total,
        int* result_observed_cloud_label,
        int cost_type,
        bool calculate_observed_cost,
        float sensor_resolution,
        float color_distance_threshold,
        float occlusion_threshold,
        std::vector<int32_t>& result_depth, 
        std::vector<std::vector<uint8_t>>& result_color,
        float* &result_cloud,
        uint8_t* &result_cloud_color,
        int& result_cloud_point_num,
        int* &result_cloud_pose_map,
        int* &result_dc_index,
        float* &rendered_cost,
        float* &observed_cost,
        float* &points_diff_cost,
        double &peak_memory_usage) {
        /*
         * - @source_mask_label - Label for every pixel in source image, used for segmentation specific occlusion checking
         * - Currently doesnt support pose occlusion or pose occlusion other
         */
        
        // std::string stage = "DEBUG";
        printf("---------------------------------------\n");
        printf("Stage : %s\n", stage.c_str());
        printf("USE_CLUTTER : %d\n", USE_CLUTTER);
        printf("USE_TREE : %d\n", USE_TREE);
        printf("sensor_resolution : %f\n", sensor_resolution);
        printf("color_distance_threshold : %f\n", color_distance_threshold);
        printf("cost_type : %d\n", cost_type);
        printf("point_dim : %d\n", point_dim);
        printf("stride : %d\n", stride);
        printf("depth_factor : %d\n", depth_factor);
        printf("observed_point_num : %d\n", observed_point_num);
        printf("occlusion_threshold : %f\n", occlusion_threshold);
        printf("calculate_observed_cost : %d\n", calculate_observed_cost);

        std::chrono::time_point<std::chrono::system_clock> start, end_1, end_2, end_3, end_3a, end_3b, end_3c, end_4;
        start = std::chrono::system_clock::now();
        // Create device inputs
        int* device_single_result_image;
        hipMalloc((void**)&device_single_result_image, sizeof(int));
        hipMemcpy(device_single_result_image, &single_result_image, sizeof(int), hipMemcpyHostToDevice);
        int num_images;
        if (single_result_image)
        {
            num_images = 1;
        }
        else
        {
            num_images = poses.size();
        }
        const Model::ROI roi= {0, 0, 0, 0};
        const size_t threadsPerBlock = 256;
        // std::cout <<tris[0].color.v1;
        thrust::device_vector<Model::Triangle> device_tris = tris;
        thrust::device_vector<Model::mat4x4> device_poses = poses;
        //// Every index maps a model id to a range of triangles in the triangle vector 
        thrust::device_vector<int> device_tris_model_count_low = tris_model_count;
        thrust::device_vector<int> device_tris_model_count_high = tris_model_count;
        thrust::device_vector<int> device_pose_model_map = pose_model_map;
        thrust::device_vector<int> device_pose_segmentation_label = pose_segmentation_label;

        thrust::device_vector<int32_t> device_source_depth = source_depth;
        thrust::device_vector<uint8_t> device_source_color_red = source_color[0];
        thrust::device_vector<uint8_t> device_source_color_green = source_color[1];
        thrust::device_vector<uint8_t> device_source_color_blue = source_color[2];
        thrust::device_vector<uint8_t> device_source_mask_label = source_mask_label;

        // thrust::copy(
        //     device_tris_model_count.begin(),
        //     device_tris_model_count.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\nPose segmentation label : \n");
        // thrust::copy(
        //     device_pose_segmentation_label.begin(),
        //     device_pose_segmentation_label.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        thrust::exclusive_scan(
            device_tris_model_count_low.begin(), device_tris_model_count_low.end(), 
            device_tris_model_count_low.begin(), 0
        ); // in-place scan
        thrust::inclusive_scan(
            device_tris_model_count_high.begin(), device_tris_model_count_high.end(), 
            device_tris_model_count_high.begin()
        ); // in-place scan
        // thrust::copy(
        //     device_tris_model_count_low.begin(),
        //     device_tris_model_count_low.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\n");
        // thrust::copy(
        //     device_tris_model_count_high.begin(),
        //     device_tris_model_count_high.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\n");
        // thrust::copy(
        //     device_pose_model_map.begin(),
        //     device_pose_model_map.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        printf("\nNumber of triangles : %d\n", tris.size());
        printf("Number of poses : %d\n", num_images);

        size_t real_width = width;
        size_t real_height = height;

        // atomic min only support int32
        
        // Create device outputs
        thrust::device_vector<int> device_pose_occluded(num_images, 0);
        thrust::device_vector<int> device_pose_occluded_other(num_images, 0);
        thrust::device_vector<float> device_pose_clutter_points(num_images, 0);
        thrust::device_vector<float> device_pose_total_points(num_images, 0);

        thrust::device_vector<int32_t> device_depth_int(num_images*real_width*real_height, INT_MAX);
        thrust::device_vector<int32_t> device_lock_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_red_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_green_int(num_images*real_width*real_height, 0);
        thrust::device_vector<uint8_t> device_blue_int(num_images*real_width*real_height, 0);

    
        Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
        Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());

        //// Mapping each pose to model
        int* device_pose_model_map_ptr = thrust::raw_pointer_cast(device_pose_model_map.data());

        //// Mapping each model to triangle range
        int* device_tris_model_count_low_ptr = thrust::raw_pointer_cast(device_tris_model_count_low.data());
        int* device_tris_model_count_high_ptr = thrust::raw_pointer_cast(device_tris_model_count_high.data());

        int* device_pose_occluded_vec = thrust::raw_pointer_cast(device_pose_occluded.data());
        int* device_pose_occluded_other_vec = thrust::raw_pointer_cast(device_pose_occluded_other.data());
        float* device_pose_clutter_points_vec = thrust::raw_pointer_cast(device_pose_clutter_points.data());
        float* device_pose_total_points_vec = thrust::raw_pointer_cast(device_pose_total_points.data());
        int* device_pose_segmentation_label_vec = thrust::raw_pointer_cast(device_pose_segmentation_label.data());
        bool use_segmentation_label = false;

        if (device_pose_segmentation_label.size() > 0)
        {
            //// 6-Dof case, segmentation label between pose and source image pixel would be compared for occlusion checking
            use_segmentation_label = true ;
        }
        printf("use_segmentation_label : %d\n", use_segmentation_label);
        int32_t* device_source_depth_vec = thrust::raw_pointer_cast(device_source_depth.data());
        uint8_t* device_source_red_vec = thrust::raw_pointer_cast(device_source_color_red.data());
        uint8_t* device_source_green_vec = thrust::raw_pointer_cast(device_source_color_green.data());
        uint8_t* device_source_blue_vec = thrust::raw_pointer_cast(device_source_color_blue.data());
        uint8_t* device_source_mask_label_vec = thrust::raw_pointer_cast(device_source_mask_label.data());

        int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());
        int32_t* lock_int_vec = thrust::raw_pointer_cast(device_lock_int.data());
        uint8_t* red_image_vec = thrust::raw_pointer_cast(device_red_int.data());
        uint8_t* green_image_vec = thrust::raw_pointer_cast(device_green_int.data());
        uint8_t* blue_image_vec = thrust::raw_pointer_cast(device_blue_int.data());
        
        peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);
        //// Render all poses
        dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
        render_triangle_multi<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                        device_poses_ptr, poses.size(),
                                                        depth_image_vec, width, height, 
                                                        device_pose_model_map_ptr, device_tris_model_count_low_ptr,
                                                        device_tris_model_count_high_ptr,
                                                        proj_mat, roi,
                                                        red_image_vec,green_image_vec,blue_image_vec,
                                                        device_source_depth_vec,
                                                        device_source_red_vec, device_source_green_vec, device_source_blue_vec,
                                                        device_pose_occluded_vec,
                                                        device_single_result_image,
                                                        lock_int_vec,
                                                        device_pose_occluded_other_vec,
                                                        device_pose_clutter_points_vec,
                                                        device_pose_total_points_vec,
                                                        device_source_mask_label_vec,
                                                        device_pose_segmentation_label_vec,
                                                        use_segmentation_label,
                                                        occlusion_threshold);

        if (USE_CLUTTER)
        {
            // printf("Pose Clutter Ratio\n");
            thrust::transform(
                device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
                device_pose_total_points.begin(), device_pose_clutter_points.begin(), 
                thrust::divides<float>()
            );
            thrust::device_vector<float> rendered_multiplier_val(num_images, 100);
            thrust::transform(
                device_pose_clutter_points.begin(), device_pose_clutter_points.end(), 
                rendered_multiplier_val.begin(), device_pose_clutter_points.begin(), 
                thrust::multiplies<float>()
            );
            thrust::copy(device_pose_clutter_points.begin(), device_pose_clutter_points.end(), clutter_cost.begin());
            // thrust::copy(
            //     device_pose_clutter_points.begin(),
            //     device_pose_clutter_points.end(), 
            //     std::ostream_iterator<float>(std::cout, " ")
            // );
            // printf("\n");
        }
        
        /// Convert INT_MAXs to zeros
        thrust::transform(device_depth_int.begin(), device_depth_int.end(), 
                            device_depth_int.begin(), max2zero_functor());
        thrust::transform(device_red_int.begin(), device_red_int.end(),
                            device_red_int.begin(), max2zero_functor());
        thrust::transform(device_green_int.begin(), device_green_int.end(),
                            device_green_int.begin(), max2zero_functor());
        thrust::transform(device_blue_int.begin(), device_blue_int.end(),
                            device_blue_int.begin(), max2zero_functor());
        
        // Free memory for stuff not needed by cloud construction
        device_tris.clear(); device_tris.shrink_to_fit();
        device_tris_model_count_low.clear(); device_tris_model_count_low.shrink_to_fit();
        device_tris_model_count_high.clear(); device_tris_model_count_high.shrink_to_fit();
        device_pose_model_map.clear(); device_pose_model_map.shrink_to_fit();
        device_poses.clear(); device_poses.shrink_to_fit();
        device_source_depth.clear(); device_source_depth.shrink_to_fit();
        device_source_color_blue.clear(); device_source_color_blue.shrink_to_fit();
        device_source_color_green.clear(); device_source_color_green.shrink_to_fit();
        device_source_color_red.clear(); device_source_color_red.shrink_to_fit();

        end_1 = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end_1-start;
        printf("*************Rendering Images Done**********\n");
        printf("*************Render time : %f*************\n", elapsed_seconds.count());
        if (stage.compare("DEBUG") == 0 || stage.compare("RENDER") == 0)
        {
            printf("Copying images to CPU\n");
            //// Allocate CPU memory
            std::vector<uint8_t> result_red(num_images*real_width*real_height);
            std::vector<uint8_t> result_green(num_images*real_width*real_height);
            std::vector<uint8_t> result_blue(num_images*real_width*real_height);
            result_depth.resize(num_images*real_width*real_height);
            
            //// Copy from GPU to CPU
            thrust::copy(device_depth_int.begin(), device_depth_int.end(), result_depth.begin());
            thrust::copy(device_red_int.begin(), device_red_int.end(), result_red.begin());
            thrust::copy(device_green_int.begin(), device_green_int.end(), result_green.begin());
            thrust::copy(device_blue_int.begin(), device_blue_int.end(), result_blue.begin());
            result_color.push_back(result_red);
            result_color.push_back(result_green);
            result_color.push_back(result_blue);

            /// Vectors will be free automatically on return

            if (stage.compare("RENDER") == 0) return;
        }
        
        ///////////////////////////////////////////////////////////////

        dim3 threadsPerBlock2D(16, 16);
        assert(real_width % stride == 0);
        dim3 numBlocks2D((real_width/stride * num_images + threadsPerBlock2D.x - 1)/threadsPerBlock2D.x, (real_height/stride + threadsPerBlock2D.y - 1)/threadsPerBlock2D.y);
        thrust::device_vector<int> mask(real_width*real_height*num_images, 0);
        int* mask_ptr = thrust::raw_pointer_cast(mask.data());

        depth_to_mask<<<numBlocks2D, threadsPerBlock2D>>>(depth_image_vec, 
                                                        mask_ptr, 
                                                        real_width, 
                                                        real_height, 
                                                        stride, 
                                                        num_images,
                                                        device_pose_occluded_vec,
                                                        NULL,
                                                        kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor,
                                                        NULL, NULL);
        if (hipGetLastError() != hipSuccess) 
        {
            printf("ERROR: Unable to execute kernel depth_to_mask\n");
        }

        //// Create mapping from pixel to corresponding index in point cloud
        int mask_back_temp = mask.back();
        thrust::exclusive_scan(mask.begin(), mask.end(), mask.begin(), 0); // in-place scan
        result_cloud_point_num = mask.back() + mask_back_temp;
        printf("Actual points in all clouds : %d\n", result_cloud_point_num);

        float* cuda_cloud;
        uint8_t* cuda_cloud_color;
        int* cuda_cloud_pose_map;
        size_t query_pitch_in_bytes;

        const unsigned int size_of_float = sizeof(float);
        const unsigned int size_of_int   = sizeof(int);
        const unsigned int size_of_uint   = sizeof(uint8_t);
        int k = 1;

        // hipMalloc(&cuda_cloud, point_dim * result_cloud_point_num * sizeof(float));
        //// Allocate memory for outputs
        hipMalloc(&cuda_cloud_color, point_dim * result_cloud_point_num * sizeof(uint8_t));
        hipMalloc(&cuda_cloud_pose_map, result_cloud_point_num * sizeof(int));
        hipMallocPitch(&cuda_cloud,   &query_pitch_in_bytes,   result_cloud_point_num * size_of_float, point_dim);

        peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);
        //// Use Mapping to convert images to point clouds
        size_t query_pitch = query_pitch_in_bytes / size_of_float;
        depth_to_2d_cloud<<<numBlocks2D, threadsPerBlock2D>>>(
                            depth_image_vec, red_image_vec, green_image_vec, blue_image_vec,
                            cuda_cloud, query_pitch_in_bytes, cuda_cloud_color, result_cloud_point_num, mask_ptr, width, height, 
                            kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor, stride, num_images, cuda_cloud_pose_map,
                            NULL, NULL, NULL, NULL);
        if (hipGetLastError() != hipSuccess) 
        {
            printf("ERROR: Unable to execute kernel depth_to_2d_cloud\n");
        }
        //// Free image memory used during point cloud construction
        device_depth_int.clear(); device_depth_int.shrink_to_fit();
        device_red_int.clear(); device_red_int.shrink_to_fit();
        device_blue_int.clear(); device_blue_int.shrink_to_fit();
        device_green_int.clear(); device_green_int.shrink_to_fit();
        if (stage.compare("DEBUG") == 0 || stage.find("CLOUD") != std::string::npos)
        {
            printf("Copying point clouds to CPU\n");
            //// Allocate CPU memory
            result_cloud = (float*) malloc(point_dim * result_cloud_point_num * sizeof(float));
            result_cloud_color = (uint8_t*) malloc(point_dim * result_cloud_point_num * sizeof(uint8_t));
            result_dc_index = (int*) malloc(num_images * width * height * sizeof(int));
            result_cloud_pose_map = (int*) malloc(result_cloud_point_num * sizeof(int));

            //// Copy to CPU if needed
            hipMemcpy2D(
                result_cloud,  result_cloud_point_num * size_of_float, cuda_cloud,  query_pitch_in_bytes,  result_cloud_point_num * size_of_float, point_dim, hipMemcpyDeviceToHost);
            // hipMemcpy(result_cloud, cuda_cloud, point_dim * result_cloud_point_num * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(result_cloud_color, cuda_cloud_color, point_dim * result_cloud_point_num * sizeof(uint8_t), hipMemcpyDeviceToHost);
            hipMemcpy(result_dc_index, mask_ptr, num_images * width * height * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(result_cloud_pose_map, cuda_cloud_pose_map, result_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
            
            /// Exit here if only point clouds are needed - for e.g. before ICP
            /// Free copied stuff
            
            if (stage.compare("CLOUD") == 0) {
                hipFree(cuda_cloud);
                hipFree(cuda_cloud_color);
                hipFree(cuda_cloud_pose_map);
                return;
            }
        }
        // Free any vectors not needed later
        mask.clear(); mask.shrink_to_fit();
        printf("************Point clouds created*************\n");
        end_2 = std::chrono::system_clock::now();
        elapsed_seconds = end_2-end_1;
        printf("************Cloud contruction time : %f************\n", elapsed_seconds.count());


        /////////////////////////////////////////////////////////////////////////////

        // Allocate memory for KNN
        // Query is render and Ref is observed
        float* ref_dev;
        float* dist_dev;
        int* index_dev;
        size_t ref_pitch_in_bytes, dist_pitch_in_bytes, index_pitch_in_bytes;
        // hipMallocPitch(&cuda_cloud, &query_pitch_in_bytes, result_cloud_point_num * size_of_float, point_dim);
        hipError_t err0, err1, err2;
        err0 = hipMallocPitch(&ref_dev, &ref_pitch_in_bytes, observed_point_num * size_of_float, point_dim);
        err1 = hipMallocPitch(&dist_dev,  &dist_pitch_in_bytes,  result_cloud_point_num * size_of_float, observed_point_num);
        err2 = hipMallocPitch(&index_dev, &index_pitch_in_bytes, result_cloud_point_num * size_of_int,   k);
        if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
            printf("ERROR: Memory allocation error (hipMallocPitch)\n");
        }
         // Deduce pitch values
        size_t ref_pitch = ref_pitch_in_bytes / size_of_float;
        size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
        size_t index_pitch = index_pitch_in_bytes / size_of_int;
        if (query_pitch != dist_pitch || query_pitch != index_pitch) {
            printf("ERROR: Invalid pitch value\n");
            return;
        }

        //// Copy observed data
        // hipMemcpy2D(cuda_cloud, query_pitch_in_bytes, result_cloud, result_cloud_point_num * size_of_float, result_cloud_point_num * size_of_float, point_dim, hipMemcpyHostToDevice);
        hipMemcpy2D(ref_dev, ref_pitch_in_bytes, observed_depth, observed_point_num * size_of_float, observed_point_num * size_of_float, point_dim, hipMemcpyHostToDevice);

        peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);
        // Compute distances and nearest neighbours
        dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
        dim3 grid0(result_cloud_point_num / BLOCK_DIM, result_cloud_point_num / BLOCK_DIM, 1);
        if (result_cloud_point_num % BLOCK_DIM != 0) grid0.x += 1;
        if (result_cloud_point_num   % BLOCK_DIM != 0) grid0.y += 1;
        compute_distances_render<<<grid0, block0>>>(ref_dev, observed_point_num, ref_pitch, cuda_cloud, result_cloud_point_num, query_pitch, point_dim, dist_dev);
        if (hipGetLastError() != hipSuccess) {
            printf("ERROR: Unable to execute kernel compute_distances_render\n");
            return;
        }
        printf("compute_distances_render() done\n");
        end_3a = std::chrono::system_clock::now();
        elapsed_seconds = end_3a-end_2;
        printf("*************compute_distances_render time : %f************\n", elapsed_seconds.count());

        dim3 block1(256, 1, 1);
        dim3 grid1(result_cloud_point_num / 256, 1, 1);
        if (result_cloud_point_num % 256 != 0) grid1.x += 1;
        modified_insertion_sort_render<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, result_cloud_point_num, observed_point_num, k);    
        if (hipGetLastError() != hipSuccess) {
            printf("ERROR: Unable to execute kernel modified_insertion_sort_render\n");
            return;
        }
        printf("modified_insertion_sort_render() done\n");
        end_3b = std::chrono::system_clock::now();
        elapsed_seconds = end_3b-end_3a;
        printf("*************modified_insertion_sort_render time : %f************\n", elapsed_seconds.count());

        dim3 block2(16, 16, 1);
        dim3 grid2(result_cloud_point_num / 16, k / 16, 1);
        if (result_cloud_point_num % 16 != 0) grid2.x += 1;
        if (k % 16 != 0)        grid2.y += 1;
        compute_sqrt_render<<<grid2, block2>>>(dist_dev, result_cloud_point_num, query_pitch, k);	
        if (hipGetLastError() != hipSuccess) {
            printf("ERROR: Unable to execute kernel compute_sqrt_render\n");
            return;
        }
        printf("compute_sqrt_render() done\n");
        end_3c = std::chrono::system_clock::now();
        elapsed_seconds = end_3c-end_3b;
        printf("*************compute_sqrt_render time : %f************\n", elapsed_seconds.count());

        // float* knn_dist;
        // int* knn_index;
        // hipMalloc(&knn_dist, result_cloud_point_num * k * size_of_float);
        // hipMalloc(&knn_index, result_cloud_point_num * k * size_of_int);

        if (stage.compare("DEBUG") == 0)
        {
            float* knn_dist_cpu   = (float*) malloc(result_cloud_point_num * k * sizeof(float));
            int* knn_index_cpu  = (int*)   malloc(result_cloud_point_num * k * sizeof(int));

            hipMemcpy2D(knn_dist_cpu,  result_cloud_point_num * size_of_float, dist_dev,  dist_pitch_in_bytes,  result_cloud_point_num * size_of_float, k, hipMemcpyDeviceToHost);
            hipMemcpy2D(knn_index_cpu, result_cloud_point_num * size_of_int,   index_dev, index_pitch_in_bytes, result_cloud_point_num * size_of_int,   k, hipMemcpyDeviceToHost);
            
            // hipMemcpy(knn_dist, knn_dist_cpu, result_cloud_point_num * size_of_float, hipMemcpyHostToDevice);
            // hipMemcpy(knn_index, knn_index_cpu, result_cloud_point_num * size_of_int, hipMemcpyHostToDevice);
            // for(int i = 0; i < result_cloud_point_num; i++){
            //     printf("knn dist:%f\n", knn_dist_cpu[i]);
            // }

            /// Not returning so need to free anything
        }
        //// Free depth point cloud and reference cloud since not needed for cost computation 
        hipFree(cuda_cloud);
        hipFree(ref_dev); //TODO
        printf("*************KNN distances computed**********\n");
        end_3 = std::chrono::system_clock::now();
        elapsed_seconds = end_3-end_2;
        printf("*************KNN time : %f************\n", elapsed_seconds.count());

        ///////////////////////////////////////////////////////////////////

        // Allocate outputs
        thrust::device_vector<float> cuda_rendered_cost_vec(num_images, 0);
        float* cuda_rendered_cost = thrust::raw_pointer_cast(cuda_rendered_cost_vec.data());
        thrust::device_vector<float> cuda_pose_point_num_vec(num_images, 0);
        float* cuda_pose_point_num = thrust::raw_pointer_cast(cuda_pose_point_num_vec.data());
        thrust::device_vector<float> cuda_rendered_explained_vec(num_images, 0);

        // Points in observed that get explained by render
        thrust::device_vector<uint8_t> cuda_observed_explained_vec(num_images * observed_point_num, 0);
        uint8_t* cuda_observed_explained = thrust::raw_pointer_cast(cuda_observed_explained_vec.data());
        int* cuda_observed_cloud_label;
        uint8_t* cuda_observed_cloud_color;

        hipMalloc(&cuda_observed_cloud_color, point_dim * observed_point_num * size_of_uint);
        hipMemcpy(cuda_observed_cloud_color, observed_color, point_dim * observed_point_num * size_of_uint, hipMemcpyHostToDevice);

        if (cost_type == 2)
        {
            hipMalloc(&cuda_observed_cloud_label, observed_point_num * size_of_int);
            hipMemcpy(cuda_observed_cloud_label, result_observed_cloud_label, observed_point_num * size_of_int, hipMemcpyHostToDevice);
        }
        peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);

        dim3 numBlocksR((result_cloud_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
        compute_render_cost<<<numBlocksR, threadsPerBlock>>>(
            dist_dev,
            index_dev,
            cuda_cloud_pose_map,
            device_pose_occluded_vec,
            cuda_rendered_cost,
            sensor_resolution,
            result_cloud_point_num,
            observed_point_num,
            cuda_pose_point_num, // Can be 0 if that pose had no points in it
            cuda_cloud_color,
            cuda_observed_cloud_color,
            cuda_cloud,
            cuda_observed_explained,
            device_pose_segmentation_label_vec,
            cuda_observed_cloud_label,
            cost_type,
            color_distance_threshold);
        
        

        thrust::device_vector<float> percentage_multiplier_val(num_images, 100);
        if (stage.compare("DEBUG") == 0 || stage.find("COST") != std::string::npos)
        {
            printf("Copying rendered cost to CPU\n");
            // Trying to get number of points explained in rendered
            thrust::transform(
                cuda_pose_point_num_vec.begin(), cuda_pose_point_num_vec.end(), 
                cuda_rendered_cost_vec.begin(), cuda_rendered_explained_vec.begin(), 
                thrust::minus<float>()
            );
            thrust::transform(
                cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
                cuda_pose_point_num_vec.begin(), cuda_rendered_cost_vec.begin(), 
                cost_percentage_functor()
                // thrust::divides<float>()
            );
            thrust::transform(
                cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
                percentage_multiplier_val.begin(), cuda_rendered_cost_vec.begin(), 
                cost_multiplier_functor()
                // thrust::multiplies<float>()
            );
            // printf("cuda_rendered_cost_vec\n");
            // std::cout << cuda_pose_point_num_vec[49] << std::endl;
            // thrust::copy(
            //     cuda_pose_point_num_vec.begin(),
            //     cuda_pose_point_num_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );
            // printf("\n");
            rendered_cost = (float*) malloc(num_images * size_of_float);
            hipMemcpy(rendered_cost, cuda_rendered_cost, num_images * size_of_float, hipMemcpyDeviceToHost);

            /// Not returning so need to free anything
        }
        printf("*************Render Costs computed**********\n");
        // if (calculate_observed_cost && cost_type == 2)
        if (calculate_observed_cost)
        {
            thrust::device_vector<float> cuda_pose_observed_explained_vec(num_images, 0);
            float* cuda_pose_observed_explained = thrust::raw_pointer_cast(cuda_pose_observed_explained_vec.data());
            thrust::device_vector<float> cuda_pose_points_diff_cost_vec(num_images, 0);

            peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);
        
            dim3 numBlocksO((num_images * observed_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
            //// Calculate the number of explained points in every pose, by adding
            compute_observed_cost<<<numBlocksO, threadsPerBlock>>>(
                num_images,
                observed_point_num,
                cuda_observed_explained,
                cuda_pose_observed_explained
            );
            
            //// Get difference of explained points between rendered and observed
            thrust::transform(
                cuda_rendered_explained_vec.begin(), cuda_rendered_explained_vec.end(), 
                cuda_pose_observed_explained_vec.begin(), cuda_pose_points_diff_cost_vec.begin(), 
                thrust::minus<float>()
            );
            // printf("Point diff\n");
            // thrust::copy(
            //     cuda_pose_points_diff_cost_vec.begin(),
            //     cuda_pose_points_diff_cost_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );
            
            // Subtract total observed points for each pose with explained points for each pose
            thrust::device_vector<float> cuda_pose_observed_points_total_vec = pose_observed_points_total;
            thrust::device_vector<float> cuda_observed_cost_vec(num_images, 0);
            thrust::transform(
                cuda_pose_observed_points_total_vec.begin(), cuda_pose_observed_points_total_vec.end(), 
                cuda_pose_observed_explained_vec.begin(), cuda_observed_cost_vec.begin(), 
                thrust::minus<float>()
            );

            // printf("Observed explained\n");
            // thrust::copy(
            //     cuda_pose_observed_points_total_vec.begin(),
            //     cuda_pose_observed_points_total_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );
            // Divide by total points
            thrust::transform(
                cuda_observed_cost_vec.begin(), cuda_observed_cost_vec.end(), 
                cuda_pose_observed_points_total_vec.begin(), cuda_observed_cost_vec.begin(), 
                thrust::divides<float>()
            );

            // Multiply by 100
            thrust::transform(
                cuda_observed_cost_vec.begin(), cuda_observed_cost_vec.end(), 
                percentage_multiplier_val.begin(), cuda_observed_cost_vec.begin(), 
                thrust::multiplies<float>()
            );

            // printf("Observed cost\n");
            // thrust::copy(
            //     cuda_observed_cost_vec.begin(),
            //     cuda_observed_cost_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );
            if (stage.compare("DEBUG") == 0 || stage.find("COST") != std::string::npos)
            {
                printf("Copying observed cost to CPU\n");
                observed_cost = (float*) malloc(num_images * size_of_float);
                points_diff_cost = (float*) malloc(num_images * size_of_float);

                float* cuda_observed_cost = thrust::raw_pointer_cast(cuda_observed_cost_vec.data());
                float* cuda_pose_points_diff_cost = thrust::raw_pointer_cast(cuda_pose_points_diff_cost_vec.data());
               
                hipMemcpy(observed_cost, cuda_observed_cost, num_images * size_of_float, hipMemcpyDeviceToHost);
                hipMemcpy(points_diff_cost, cuda_pose_points_diff_cost, num_images * size_of_float, hipMemcpyDeviceToHost);

                /// Not returning so need to free anything
            }
        }
        // else if (calculate_observed_cost && cost_type != 2)
        // {
        //     thrust::device_vector<float> cuda_pose_observed_unexplained_vec(num_images, 0);
        //     float* cuda_pose_observed_unexplained = thrust::raw_pointer_cast(cuda_pose_observed_unexplained_vec.data());
        //     peak_memory_usage = std::max(print_cuda_memory_usage(), peak_memory_usage);
        
        //     thrust::device_vector<float> device_poses_cylinder_radius_vec(num_images, 0.5);
        //     float* device_poses_cylinder_radius_ptr = thrust::raw_pointer_cast(device_poses_cylinder_radius_vec.data());
            
        //     dim3 numBlocksO((num_images * observed_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
        //     //// Calculate the number of explained points in every pose, by adding
        //     compute_observed_cost_cylinder<<<numBlocksO, threadsPerBlock>>>(
        //         num_images,
        //         observed_point_num,
        //         cuda_observed_explained,
        //         ref_dev,
        //         ref_pitch_in_bytes,
        //         device_poses_ptr,
        //         device_poses_cylinder_radius_ptr,
        //         cuda_pose_observed_unexplained
        //     );
        //     thrust::copy(
        //         cuda_pose_observed_unexplained_vec.begin(),
        //         cuda_pose_observed_unexplained_vec.end(), 
        //         std::ostream_iterator<float>(std::cout, " ")
        //     );
        //     printf("\n");
        //     // if (stage.compare("DEBUG") == 0 || stage.compare("COST") == 0)
        //     // {
        //     //     printf("Copying observed cost to CPU\n");
        //     //     observed_cost = (float*) malloc(num_images * size_of_float);

        //     //     float* cuda_observed_cost = thrust::raw_pointer_cast(cuda_observed_cost_vec.data());
               
        //     //     hipMemcpy(observed_cost, cuda_observed_cost, num_images * size_of_float, hipMemcpyDeviceToHost);

        //     //     /// Not returning so need to free anything
        //     // }

        // }

        
        hipFree(cuda_cloud_color);
        hipFree(cuda_cloud_pose_map);
        hipFree(index_dev);
        hipFree(dist_dev);
        if (cost_type == 2) hipFree(cuda_observed_cloud_label);
        hipFree(cuda_observed_cloud_color);

        end_4 = std::chrono::system_clock::now();
        elapsed_seconds = end_4-end_3;
        printf("************Cost Computation time : %f************\n", elapsed_seconds.count());

        printf("---------------------------------------\n");

    }
    struct concatenate_transforms{

        concatenate_transforms(){}

        __host__ __device__
        Model::mat4x4 operator()(Model::mat4x4& transform_a_mat, Eigen::Isometry3f& transform_b) const
        {
            // printf("Before icp : \n");
            // transform_a_mat.print_device();
            Eigen::Matrix4f transform_a = transform_a_mat.to_eigen(100);
            Eigen::Matrix4f transform_new = transform_b.matrix() * transform_a;
            Model::mat4x4 transform_new_mat;
            transform_new_mat.init_from_eigen(transform_new.cast<double>(), 100);
            // printf("After icp : \n");
            // transform_new_mat.print_device();
            return transform_new_mat;
        }
    };

    void render_cuda_multi_unified(
        const std::string stage,
        const std::vector<Model::Triangle>& tris,
        const std::vector<Model::mat4x4>& poses,
        const std::vector<int> pose_model_map,
        const std::vector<int> tris_model_count,
        size_t width, size_t height, const Model::mat4x4& proj_mat,
        const std::vector<int32_t>& source_depth,
        const std::vector<std::vector<uint8_t>>& source_color,
        int single_result_image,
        std::vector<float>& clutter_cost,
        const std::vector<uint8_t>& source_mask_label,
        const std::vector<int>& pose_segmentation_label,
        int stride,
        int point_dim,
        int depth_factor,
        float kCameraCX,
        float kCameraCY,
        float kCameraFX,
        float kCameraFY,
        float* observed_depth,
        Eigen::Vector3f* observed_depth_eigen,
        uint8_t* observed_color,
        int observed_point_num,
        std::vector<float> pose_observed_points_total,
        int* result_observed_cloud_label,
        int cost_type,
        bool calculate_observed_cost,
        float sensor_resolution,
        float color_distance_threshold,
        float occlusion_threshold,
        bool do_icp,
        std::vector<int32_t>& result_depth, 
        std::vector<std::vector<uint8_t>>& result_color,
        float* &result_cloud,
        uint8_t* &result_cloud_color,
        int& result_cloud_point_num,
        int* &result_cloud_pose_map,
        int* &result_dc_index,
        std::vector<Model::mat4x4>& adjusted_poses,
        float* &rendered_cost,
        float* &observed_cost,
        float* &points_diff_cost,
        gpu_stats& stats) {
        /*
         * - @source_mask_label - Label for every pixel in source image, used for segmentation specific occlusion checking
         * - Currently doesnt support pose occlusion or pose occlusion other
         */
        
        // std::string stage = "DEBUG";
        // gpu_stats stats;
        printf("---------------------------------------\n");
        printf("Stage : %s\n", stage.c_str());
        printf("USE_CLUTTER : %d\n", USE_CLUTTER);
        printf("USE_TREE : %d\n", USE_TREE);
        printf("sensor_resolution : %f\n", sensor_resolution);
        printf("color_distance_threshold : %f\n", color_distance_threshold);
        printf("cost_type : %d\n", cost_type);
        printf("point_dim : %d\n", point_dim);
        printf("stride : %d\n", stride);
        printf("depth_factor : %d\n", depth_factor);
        printf("observed_point_num : %d\n", observed_point_num);
        printf("occlusion_threshold : %f\n", occlusion_threshold);
        printf("calculate_observed_cost : %d\n", calculate_observed_cost);

        std::chrono::time_point<std::chrono::system_clock> start, end_1, end_2, end_3, end_3a, end_3b, end_3c, end_4;
        start = std::chrono::system_clock::now();
        // Create device inputs
        // int* device_single_result_image;
        // hipMalloc((void**)&device_single_result_image, sizeof(int));
        // hipMemcpy(device_single_result_image, &single_result_image, sizeof(int), hipMemcpyHostToDevice);
        int num_images;
        if (single_result_image)
        {
            num_images = 1;
        }
        else
        {
            num_images = poses.size();
        }
        // const Model::ROI roi= {0, 0, 0, 0};
        const size_t threadsPerBlock = 256;
        // std::cout <<tris[0].color.v1;
        thrust::device_vector<Model::Triangle> device_tris = tris;
        thrust::device_vector<Model::mat4x4> device_poses = poses;
        //// Every index maps a model id to a range of triangles in the triangle vector 
        thrust::device_vector<int> device_tris_model_count_low = tris_model_count;
        thrust::device_vector<int> device_tris_model_count_high = tris_model_count;
        thrust::device_vector<int> device_pose_model_map = pose_model_map;
        thrust::device_vector<int> device_pose_segmentation_label = pose_segmentation_label;

        thrust::device_vector<int32_t> device_source_depth = source_depth;
        thrust::device_vector<uint8_t> device_source_color_red = source_color[0];
        thrust::device_vector<uint8_t> device_source_color_green = source_color[1];
        thrust::device_vector<uint8_t> device_source_color_blue = source_color[2];
        thrust::device_vector<uint8_t> device_source_mask_label = source_mask_label;

        size_t real_width = width;
        size_t real_height = height;

        // Create device outputs
        thrust::device_vector<int> device_pose_occluded;
        thrust::device_vector<int> device_pose_occluded_other;
        thrust::device_vector<float> device_pose_clutter_points;
        thrust::device_vector<float> device_pose_total_points;

        thrust::device_vector<int32_t> device_depth_int;
        thrust::device_vector<int32_t> device_lock_int;
        thrust::device_vector<uint8_t> device_red_int;
        thrust::device_vector<uint8_t> device_green_int;
        thrust::device_vector<uint8_t> device_blue_int;
        image_render(device_tris,
                    device_poses,
                    device_pose_model_map,
                    device_tris_model_count_low,
                    device_source_depth,
                    device_source_color_red,
                    device_source_color_green,
                    device_source_color_blue,
                    device_source_mask_label,
                    device_pose_segmentation_label,
                    num_images,
                    width,
                    height,
                    proj_mat,
                    occlusion_threshold,
                    single_result_image,
                    device_pose_occluded,
                    device_pose_occluded_other,
                    device_pose_clutter_points,
                    device_pose_total_points,
                    device_depth_int,
                    device_red_int,
                    device_green_int,
                    device_blue_int,
                    stats);
        // int* device_pose_occluded_vec = thrust::raw_pointer_cast(device_pose_occluded.data());
        // int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());
        // uint8_t* red_image_vec = thrust::raw_pointer_cast(device_red_int.data());
        // uint8_t* green_image_vec = thrust::raw_pointer_cast(device_green_int.data());
        // uint8_t* blue_image_vec = thrust::raw_pointer_cast(device_blue_int.data());
        // int* device_pose_segmentation_label_vec = thrust::raw_pointer_cast(device_pose_segmentation_label.data());
        if (USE_CLUTTER) {
            thrust::copy(device_pose_clutter_points.begin(), device_pose_clutter_points.end(), clutter_cost.begin());
        }
        end_1 = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end_1-start;
        printf("*************Rendering Images Done**********\n");
        printf("*************Render time : %f*************\n", elapsed_seconds.count());
        if (stage.compare("DEBUG") == 0 || stage.compare("RENDER") == 0)
        {
            printf("Copying images to CPU\n");
            //// Allocate CPU memory
            std::vector<uint8_t> result_red(num_images*real_width*real_height);
            std::vector<uint8_t> result_green(num_images*real_width*real_height);
            std::vector<uint8_t> result_blue(num_images*real_width*real_height);
            result_depth.resize(num_images*real_width*real_height);
            
            //// Copy from GPU to CPU
            thrust::copy(device_depth_int.begin(), device_depth_int.end(), result_depth.begin());
            thrust::copy(device_red_int.begin(), device_red_int.end(), result_red.begin());
            thrust::copy(device_green_int.begin(), device_green_int.end(), result_green.begin());
            thrust::copy(device_blue_int.begin(), device_blue_int.end(), result_blue.begin());
            result_color.push_back(result_red);
            result_color.push_back(result_green);
            result_color.push_back(result_blue);

            /// Vectors will be free automatically on return

            if (stage.compare("RENDER") == 0) return;
        }
        ///////////////////////////////////////////////////////////////
        const unsigned int size_of_float = sizeof(float);
        const unsigned int size_of_int   = sizeof(int);
        const unsigned int size_of_uint   = sizeof(uint8_t);
        float* cuda_cloud;
        size_t query_pitch_in_bytes;
        // int rendered_cloud_point_num;
        thrust::device_vector<float>   rendered_point_cloud;
        thrust::device_vector<uint8_t> rendered_point_cloud_color;
        thrust::device_vector<int>     rendered_dc_index;
        thrust::device_vector<int>     rendered_cloud_pose_map;
        thrust::device_vector<int>     dummy_vec;
        thrust::device_vector<Eigen::Vector3f> result_cloud_eigen;
        compute_point_clouds(
            device_depth_int,
            device_red_int,
            device_green_int,
            device_blue_int,
            num_images,
            width,
            height,
            kCameraCX,
            kCameraCY,
            kCameraFX,
            kCameraFY,
            depth_factor,
            stride,
            device_pose_occluded,
            cuda_cloud,
            query_pitch_in_bytes,
            result_cloud_eigen,
            rendered_point_cloud,
            rendered_point_cloud_color,
            result_cloud_point_num,
            rendered_dc_index,
            rendered_cloud_pose_map,
            dummy_vec,
            stats
        );
        int k = 1;
        printf("************Point clouds created*************\n");
        end_2 = std::chrono::system_clock::now();
        elapsed_seconds = end_2-end_1;
        printf("************Cloud contruction time : %f************\n", elapsed_seconds.count());
        
        // Copy observed stuff from CPU
        thrust::device_vector<int> observed_cloud_label(result_observed_cloud_label, result_observed_cloud_label + observed_point_num);
        thrust::device_vector<Eigen::Vector3f> observed_cloud_eigen(observed_depth_eigen, observed_depth_eigen + observed_point_num);
        printf("observed_cloud_eigen() size : %d\n", observed_cloud_eigen.size());

        // ///////////////////////////////////////////////////////////////
        // // Testing ICP
        end_3a = end_2;
        if (do_icp)
        {
            thrust::device_vector<int> observed_cloud_label_subtracted(observed_cloud_label.size());
            thrust::device_vector<int> device_pose_segmentation_label_subtracted(device_pose_segmentation_label.size());
            if (device_pose_segmentation_label.size() > 0)
            {
                printf("Subtracting 1 from segmentation labels for ICP\n");
                // Make segmentation labes start from 0
                thrust::device_vector<int> minus_vec(observed_cloud_label.size(), 1);
                thrust::transform(
                    observed_cloud_label.begin(), observed_cloud_label.end(), 
                    minus_vec.begin(), observed_cloud_label_subtracted.begin(), 
                    thrust::minus<float>()
                );

                minus_vec.resize(device_pose_segmentation_label.size(), 1);
                thrust::transform(
                    device_pose_segmentation_label.begin(), device_pose_segmentation_label.end(), 
                    minus_vec.begin(), device_pose_segmentation_label_subtracted.begin(), 
                    thrust::minus<float>()
                );
            }


            std::vector<Eigen::Isometry3f> estimated;
            thrust::device_vector<Eigen::Isometry3f> d_estimated;
            std::unique_ptr<fast_gicp::FastGICPCudaCore> vgicp_cuda;
            int max_iterations_ = 150;
            int k_correspondences_ = 10;
            float rotation_epsilon_ = 2e-3;
            float transformation_epsilon_ = 5e-4;
                
            vgicp_cuda.reset(new fast_gicp::FastGICPCudaCore());
            vgicp_cuda->set_max_iterations(max_iterations_);
            vgicp_cuda->set_rotation_epsilon(rotation_epsilon_);
            vgicp_cuda->set_transformation_epsilon(transformation_epsilon_);
            vgicp_cuda->set_correspondence_randomness(k_correspondences_);
            vgicp_cuda->set_input(result_cloud_eigen,
                                observed_cloud_eigen,
                                rendered_cloud_pose_map,
                                observed_cloud_label_subtracted,
                                device_pose_segmentation_label_subtracted,
                                num_images);
            vgicp_cuda->optimize_multi(estimated);
            d_estimated = estimated;
            thrust::device_vector<Model::mat4x4> device_poses_adjusted(device_poses.size());
            thrust::host_vector<Model::mat4x4> host_poses_adjusted(device_poses.size());
            thrust::transform(
                device_poses.begin(), device_poses.end(), d_estimated.begin(), device_poses_adjusted.begin(),
                concatenate_transforms()
            );
            adjusted_poses.resize(num_images);
            thrust::copy(device_poses_adjusted.begin(), device_poses_adjusted.end(), adjusted_poses.begin());
            // host_poses_adjusted = device_poses_adjusted;
            // for (int i = 0; i < host_poses_adjusted.size(); i++)
            // {
            //     host_poses_adjusted[i].print();
            // }
            printf("*************ICPs computed**********\n");
            end_3a = std::chrono::system_clock::now();
            elapsed_seconds = end_3a-end_2;
            printf("*************ICP time : %f************\n", elapsed_seconds.count());
            stats.icp_runtime = std::max(stats.icp_runtime, (float) elapsed_seconds.count());

            image_render(device_tris,
                device_poses_adjusted,
                device_pose_model_map,
                device_tris_model_count_low,
                device_source_depth,
                device_source_color_red,
                device_source_color_green,
                device_source_color_blue,
                device_source_mask_label,
                device_pose_segmentation_label,
                num_images,
                width,
                height,
                proj_mat,
                occlusion_threshold,
                single_result_image,
                device_pose_occluded,
                device_pose_occluded_other,
                device_pose_clutter_points,
                device_pose_total_points,
                device_depth_int,
                device_red_int,
                device_green_int,
                device_blue_int,
                stats);

            compute_point_clouds(
                device_depth_int,
                device_red_int,
                device_green_int,
                device_blue_int,
                num_images,
                width,
                height,
                kCameraCX,
                kCameraCY,
                kCameraFX,
                kCameraFY,
                depth_factor,
                stride,
                device_pose_occluded,
                cuda_cloud,
                query_pitch_in_bytes,
                result_cloud_eigen,
                rendered_point_cloud,
                rendered_point_cloud_color,
                result_cloud_point_num,
                rendered_dc_index,
                rendered_cloud_pose_map,
                dummy_vec,
                stats
            );
        }
        
        ///////////////////////////////////////////////////////////////

        if (stage.compare("DEBUG") == 0 || stage.find("CLOUD") != std::string::npos)
        {
            printf("Copying point clouds to CPU\n");
            //// Allocate CPU memory
            result_cloud = (float*) malloc(point_dim * result_cloud_point_num * sizeof(float));
            result_cloud_color = (uint8_t*) malloc(point_dim * result_cloud_point_num * sizeof(uint8_t));
            result_dc_index = (int*) malloc(num_images * width * height * sizeof(int));
            result_cloud_pose_map = (int*) malloc(result_cloud_point_num * sizeof(int));

            //// Copy to CPU if needed
            hipMemcpy2D(
                result_cloud,  result_cloud_point_num * size_of_float, cuda_cloud,  query_pitch_in_bytes,  result_cloud_point_num * size_of_float, point_dim, hipMemcpyDeviceToHost);
            // hipMemcpy(result_cloud, cuda_cloud, point_dim * result_cloud_point_num * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(result_cloud_color, thrust::raw_pointer_cast(rendered_point_cloud_color.data()), point_dim * result_cloud_point_num * sizeof(uint8_t), hipMemcpyDeviceToHost);
            hipMemcpy(result_dc_index, thrust::raw_pointer_cast(rendered_dc_index.data()), num_images * width * height * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(result_cloud_pose_map, thrust::raw_pointer_cast(rendered_cloud_pose_map.data()), result_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
            
            /// Exit here if only point clouds are needed - for e.g. before ICP
            /// Free copied stuff
            
            if (stage.compare("CLOUD") == 0) {
                hipFree(cuda_cloud);
                // hipFree(cuda_cloud_color);
                // hipFree(cuda_cloud_pose_map);
                return;
            }
        }
        rendered_dc_index.clear(); rendered_dc_index.shrink_to_fit();
        ///////////////////////////////////////////////////////////////
        
        thrust::device_vector<thrust::pair<float, int>> k_neighbors;
        fast_gicp::brute_force_knn_search(result_cloud_eigen, 
                                        observed_cloud_eigen, 
                                        1, 
                                        k_neighbors);
                                        //// thrust::device_vector<int>(0), // NN will not be segmentation specific
                                        //// thrust::device_vector<int>(0), // NN will not be segmentation specific
                                        //*source_label_map,
                                        //*target_label_indices,
                                        //*source_pose_map,
                                        //adjusted_x0s,
                                        //mask_pose_icp);
        printf("*************KNN distances computed**********\n");
        end_3 = std::chrono::system_clock::now();
        elapsed_seconds = end_3-end_3a;
        printf("*************KNN time : %f************\n", elapsed_seconds.count());
        sensor_resolution = sensor_resolution * sensor_resolution;
               
        thrust::device_vector<float> k_distances(k_neighbors.size());
        thrust::device_vector<int> k_indices(k_neighbors.size());
        thrust::transform(k_neighbors.begin(), k_neighbors.end(), k_indices.begin(), fast_gicp::untie_pair_second());
        thrust::transform(k_neighbors.begin(), k_neighbors.end(), k_distances.begin(), fast_gicp::untie_pair_first());
        //////////////////////////////////////////////////////////////

        // Testing new cost compute interface
        thrust::device_vector<uint8_t> observed_cloud_color(observed_color, observed_color + point_dim * observed_point_num);

        thrust::device_vector<float> rendered_poses_observed_points_total = pose_observed_points_total;
        thrust::device_vector<float> cuda_rendered_cost_v;
        thrust::device_vector<float> cuda_observed_cost_v;
        thrust::device_vector<float> cuda_pose_points_diff_cost_vec;
        compute_costs(num_images,
            cost_type,
            calculate_observed_cost,
            sensor_resolution,
            color_distance_threshold,
            observed_cloud_color,
            observed_cloud_label,
            observed_point_num,
            rendered_point_cloud_color,
            rendered_cloud_pose_map,
            device_pose_occluded,
            device_pose_segmentation_label,
            rendered_poses_observed_points_total,
            result_cloud_point_num,
            k_distances,
            k_indices,
            cuda_rendered_cost_v,
            cuda_observed_cost_v,
            cuda_pose_points_diff_cost_vec,
            stats                        
        );
        if (stage.compare("DEBUG") == 0 || stage.find("COST") != std::string::npos)
        {
            printf("Copying rendered cost to CPU\n");
            rendered_cost = (float*) malloc(num_images * size_of_float);
            hipMemcpy(rendered_cost, thrust::raw_pointer_cast(cuda_rendered_cost_v.data()), num_images * size_of_float, hipMemcpyDeviceToHost);
            if (calculate_observed_cost)
            {
                printf("Copying observed cost to CPU\n");
                observed_cost = (float*) malloc(num_images * size_of_float);
                points_diff_cost = (float*) malloc(num_images * size_of_float);

                hipMemcpy(observed_cost, thrust::raw_pointer_cast(cuda_observed_cost_v.data()), num_images * size_of_float, hipMemcpyDeviceToHost);
                hipMemcpy(points_diff_cost, thrust::raw_pointer_cast(cuda_pose_points_diff_cost_vec.data()), num_images * size_of_float, hipMemcpyDeviceToHost);
            }
        }
        // hipFree(index_dev);
        // hipFree(dist_dev);
        // if (cost_type == 2) hipFree(cuda_observed_cloud_label);
        // hipFree(cuda_observed_cloud_color);

        end_4 = std::chrono::system_clock::now();
        elapsed_seconds = end_4-end_3;
        printf("*************Costs computed**********\n");
        printf("************Cost Computation time : %f************\n", elapsed_seconds.count());
        return;
        ///////////////////////////////////////////////////////////////////
    }

    bool depth2cloud_global(const std::vector<int32_t>& depth_data,
                            const std::vector<std::vector<uint8_t>>& color_data,
                            Eigen::Vector3f* &result_cloud_eigen,
                            float* &result_cloud,
                            uint8_t* &result_cloud_color,
                            int* &dc_index,
                            int &rendered_cloud_point_num,
                            int* &cloud_pose_map,
                            int* &result_observed_cloud_label,
                            const int width, 
                            const int height, 
                            const int num_poses,
                            const std::vector<int>& pose_occluded,
                            const float kCameraCX, 
                            const float kCameraCY, 
                            const float kCameraFX, 
                            const float kCameraFY,
                            const float depth_factor,
                            const int stride,
                            const int point_dim,
                            const std::vector<uint8_t>& label_mask_data,
                            const std::vector<double>& observed_cloud_bounds,
                            const Eigen::Matrix4f* camera_transform)
    {
        printf("depth2cloud_global()\n");
        /**
            Convert a given input to point cloud, used to convert observed images to point cloud
            @label_mask_data - Label of every pixel in input 2D image
            @result_observed_cloud_label - Label of every pixel in output 3D cloud, downsampled
            Returns :
                cloud_pose_map - Mapping of every point in the cloud to a pose number
                rendered_cloud_point_num - Total number of points in the rendered pose arrays
        */

        //// Testing new interface
        thrust::device_vector<uint8_t> d_red_in = color_data[0];
        thrust::device_vector<uint8_t> d_green_in = color_data[1];
        thrust::device_vector<uint8_t> d_blue_in = color_data[2];

        thrust::device_vector<int32_t> d_depth_data = depth_data;
        thrust::device_vector<int> d_poses_occluded = pose_occluded;
        thrust::device_vector<uint8_t> d_label_mask_data = label_mask_data;
        thrust::device_vector<double> d_observed_cloud_bounds = observed_cloud_bounds;
        thrust::device_vector<float>   result_point_cloud;
        thrust::device_vector<uint8_t> result_point_cloud_color;
        thrust::device_vector<int>     result_dc_index;
        thrust::device_vector<int>     result_cloud_pose_map;
        thrust::device_vector<int>     result_cloud_label;
        thrust::device_vector<Eigen::Vector3f> d_result_cloud_eigen;

        Eigen::Matrix4f* d_camera_transform;
        if (camera_transform != NULL)
        {
            printf("Using camera transform to transform point cloud to world frame\n");
            std::cout << *camera_transform << std::endl; 
            hipMalloc(&d_camera_transform, sizeof(Eigen::Matrix4f));
            hipMemcpy(d_camera_transform, camera_transform, sizeof(Eigen::Matrix4f), hipMemcpyHostToDevice);
        }

        float* result_2d_point_cloud;
        size_t result_2d_pitch_in_bytes;
        gpu_stats stats;
        compute_point_clouds(
            d_depth_data,
            d_red_in,
            d_green_in,
            d_blue_in,
            num_poses,
            width,
            height,
            kCameraCX,
            kCameraCY,
            kCameraFX,
            kCameraFY,
            depth_factor,
            stride,
            d_poses_occluded,
            result_2d_point_cloud,
            result_2d_pitch_in_bytes,
            d_result_cloud_eigen,
            result_point_cloud,
            result_point_cloud_color,
            rendered_cloud_point_num,
            result_dc_index,
            result_cloud_pose_map,
            result_cloud_label,
            stats,
            d_camera_transform,
            d_label_mask_data,
            d_observed_cloud_bounds
        );
        // thrust::copy(
        //     result_cloud_label.begin(),
        //     result_cloud_label.end(), 
        //     std::ostream_iterator<int>(std::cout, " ")
        // );
        // printf("\n");
        // std::vector<float> c_point_cloud(result_point_cloud.size());
        // std::vector<uint8_t> c_point_cloud_color(result_point_cloud_color.size());
        // std::vector<int> c_dc_index(result_dc_index.size());
        // std::vector<int> d_cloud_pose_map(result_cloud_pose_map.size());
        // std::vector<int> d_cloud_label(result_cloud_label.size());
        // thrust::copy(result_point_cloud.begin(), result_point_cloud.end(), c_point_cloud.begin());
        // thrust::copy(result_point_cloud_color.begin(), result_point_cloud_color.end(), c_point_cloud_color.begin());
        // thrust::copy(result_dc_index.begin(), result_dc_index.end(), c_dc_index.begin());
        // thrust::copy(result_cloud_pose_map.begin(), result_cloud_pose_map.end(), d_cloud_pose_map.begin());
        // thrust::copy(result_cloud_label.begin(), result_cloud_label.end(), d_cloud_label.begin());

        // Assign host pointers
        result_cloud = (float*) malloc(point_dim * rendered_cloud_point_num * sizeof(float));
        result_cloud_eigen = (Eigen::Vector3f*) malloc(rendered_cloud_point_num * sizeof(Eigen::Vector3f));
        result_cloud_color = (uint8_t*) malloc(point_dim * rendered_cloud_point_num * sizeof(uint8_t));
        dc_index = (int*) malloc(num_poses * width * height * sizeof(int));
        cloud_pose_map = (int*) malloc(rendered_cloud_point_num * sizeof(int));
        result_observed_cloud_label = (int*) malloc(rendered_cloud_point_num * sizeof(int));

        hipMemcpy2D(
            result_cloud,  rendered_cloud_point_num * sizeof(float), result_2d_point_cloud,  result_2d_pitch_in_bytes,  rendered_cloud_point_num * sizeof(float), point_dim, hipMemcpyDeviceToHost);

        // hipMemcpy(result_cloud, thrust::raw_pointer_cast(result_point_cloud.data()), point_dim * rendered_cloud_point_num * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(result_cloud_eigen, thrust::raw_pointer_cast(d_result_cloud_eigen.data()), rendered_cloud_point_num * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost);
        hipMemcpy(result_cloud_color, thrust::raw_pointer_cast(result_point_cloud_color.data()), point_dim * rendered_cloud_point_num * sizeof(uint8_t), hipMemcpyDeviceToHost);
        hipMemcpy(dc_index, thrust::raw_pointer_cast(result_dc_index.data()), num_poses * width * height * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(cloud_pose_map, thrust::raw_pointer_cast(result_cloud_pose_map.data()), rendered_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
        if (label_mask_data.size() > 0)
        {
            hipMemcpy(result_observed_cloud_label, thrust::raw_pointer_cast(result_cloud_label.data()), rendered_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
        }
        if (camera_transform != NULL)
        {
            hipFree(d_camera_transform);
        }
        hipFree(result_2d_point_cloud);
        return true;
        
        // thrust::device_vector<uint8_t> d_red_in = color_data[0];
        // thrust::device_vector<uint8_t> d_green_in = color_data[1];
        // thrust::device_vector<uint8_t> d_blue_in = color_data[2];
        // int32_t* depth_data_cuda;
        // int* pose_occluded_cuda;
        // uint8_t* label_mask_data_cuda = NULL;
        // double* observed_cloud_bounds_cuda = NULL;
        // Eigen::Matrix4f* camera_transform_cuda = NULL;
        // // int stride = 5;
        // hipMalloc(&depth_data_cuda, num_poses * width * height * sizeof(int32_t));
        // hipMemcpy(depth_data_cuda, depth_data.data(), num_poses * width * height * sizeof(int32_t), hipMemcpyHostToDevice);
        
        // if (label_mask_data.size() > 0)
        // {
        //     printf("Using segementation labels to create point cloud\n");
        //     hipMalloc(&label_mask_data_cuda, num_poses * width * height * sizeof(uint8_t));
        //     hipMemcpy(label_mask_data_cuda, label_mask_data.data(), num_poses * width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
        // }
        // if (observed_cloud_bounds.size() > 0)
        // {
        //     // std::cout << observed_cloud_bounds[0] << " " << observed_cloud_bounds[1] << std::endl;
        //     printf("Using filter bounds to create point cloud\n");
        //     printf("x_min : %f, x_max : %f\n", observed_cloud_bounds[1], observed_cloud_bounds[0]);
        //     printf("y_min : %f, y_max : %f\n", observed_cloud_bounds[3], observed_cloud_bounds[2]);
        //     printf("z_min : %f, z_max : %f\n", observed_cloud_bounds[5], observed_cloud_bounds[4]);
        //     hipMalloc(&observed_cloud_bounds_cuda, 6 * sizeof(double));
        //     hipMemcpy(observed_cloud_bounds_cuda, observed_cloud_bounds.data(), 6 * sizeof(double), hipMemcpyHostToDevice);
        // }
        // if (camera_transform != NULL)
        // {
        //     printf("Using camera transform to transform point cloud to world frame\n");
        //     std::cout << *camera_transform << std::endl; 
        //     hipMalloc(&camera_transform_cuda, sizeof(Eigen::Matrix4f));
        //     hipMemcpy(camera_transform_cuda, camera_transform, sizeof(Eigen::Matrix4f), hipMemcpyHostToDevice);
        // }
        // hipMalloc(&pose_occluded_cuda, num_poses * sizeof(int));
        // hipMemcpy(pose_occluded_cuda, pose_occluded.data(), num_poses * sizeof(int), hipMemcpyHostToDevice);

        // dim3 threadsPerBlock(16, 16);
        // assert(width % stride == 0);
        // dim3 numBlocks((width/stride * num_poses + threadsPerBlock.x - 1)/threadsPerBlock.x, (height/stride + threadsPerBlock.y - 1)/threadsPerBlock.y);

        // thrust::device_vector<int> mask(width*height*num_poses, 0);
        // int* mask_ptr = thrust::raw_pointer_cast(mask.data());

        // depth_to_mask<<<numBlocks, threadsPerBlock>>>(depth_data_cuda, 
        //                                               mask_ptr, width, 
        //                                               height, 
        //                                               stride, 
        //                                               num_poses,
        //                                               pose_occluded_cuda,
        //                                               label_mask_data_cuda,
        //                                               kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor,
        //                                               observed_cloud_bounds_cuda, camera_transform_cuda);
            
        // if (hipGetLastError() != hipSuccess) 
        // {
        //     printf("ERROR: Unable to execute kernel depth_to_mask\n");
        //     return false;
        // }
        // // hipDeviceSynchronize();

        // // Create mapping from pixel to corresponding index in point cloud
        // int mask_back_temp = mask.back();
        // thrust::exclusive_scan(mask.begin(), mask.end(), mask.begin(), 0); // in-place scan
        // rendered_cloud_point_num = mask.back() + mask_back_temp;
        // printf("Actual points in all clouds : %d\n", rendered_cloud_point_num);

        // float* cuda_cloud;
        // uint8_t* cuda_cloud_color;
        // int* cuda_cloud_pose_map;
        // int* cuda_cloud_mask_label;
        // size_t query_pitch_in_bytes;

        // // hipMalloc(&cuda_cloud, point_dim * rendered_cloud_point_num * sizeof(float));
        // hipMallocPitch(&cuda_cloud,   &query_pitch_in_bytes,   rendered_cloud_point_num * sizeof(float), point_dim);
        // hipMalloc(&cuda_cloud_color, point_dim * rendered_cloud_point_num * sizeof(uint8_t));
        // hipMalloc(&cuda_cloud_pose_map, rendered_cloud_point_num * sizeof(int));
        // if (label_mask_data.size() > 0)
        // {
        //     hipMalloc(&cuda_cloud_mask_label, rendered_cloud_point_num * sizeof(int));
        // }

        // result_cloud = (float*) malloc(point_dim * rendered_cloud_point_num * sizeof(float));
        // result_cloud_color = (uint8_t*) malloc(point_dim * rendered_cloud_point_num * sizeof(uint8_t));
        // dc_index = (int*) malloc(num_poses * width * height * sizeof(int));
        // cloud_pose_map = (int*) malloc(rendered_cloud_point_num * sizeof(int));
        // result_observed_cloud_label = (int*) malloc(rendered_cloud_point_num * sizeof(int));

        

        // uint8_t* red_in = thrust::raw_pointer_cast(d_red_in.data());
        // uint8_t* green_in = thrust::raw_pointer_cast(d_green_in.data());
        // uint8_t* blue_in = thrust::raw_pointer_cast(d_blue_in.data());

        // depth_to_2d_cloud<<<numBlocks, threadsPerBlock>>>(
        //                     depth_data_cuda, red_in, green_in, blue_in,
        //                     cuda_cloud, query_pitch_in_bytes, cuda_cloud_color, rendered_cloud_point_num, mask_ptr, width, height, 
        //                     kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor, stride, num_poses, cuda_cloud_pose_map,
        //                     label_mask_data_cuda, cuda_cloud_mask_label, observed_cloud_bounds_cuda, camera_transform_cuda);
        // // depth_to_cloud<<<numBlocks, threadsPerBlock>>>(
        // //                     depth_data_cuda, red_in, green_in, blue_in,
        // //                     cuda_cloud, cuda_cloud_color, rendered_cloud_point_num, mask_ptr, width, height, 
        // //                     kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor, stride, cuda_cloud_pose_map,
        // //                     label_mask_data_cuda, cuda_cloud_mask_label);
        // if (hipGetLastError() != hipSuccess) 
        // {
        //     printf("ERROR: Unable to execute kernel depth_to_2d_cloud\n");
        //     return false;
        // }
        // // hipDeviceSynchronize();
        // hipMemcpy2D(
        //         result_cloud,  rendered_cloud_point_num * sizeof(float), cuda_cloud,  query_pitch_in_bytes,  rendered_cloud_point_num * sizeof(float), point_dim, hipMemcpyDeviceToHost);
        // // hipMemcpy(result_cloud, cuda_cloud, point_dim * rendered_cloud_point_num * sizeof(float), hipMemcpyDeviceToHost);
        // hipMemcpy(result_cloud_color, cuda_cloud_color, point_dim * rendered_cloud_point_num * sizeof(uint8_t), hipMemcpyDeviceToHost);
        // hipMemcpy(dc_index, mask_ptr, num_poses * width * height * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(cloud_pose_map, cuda_cloud_pose_map, rendered_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
        // if (label_mask_data.size() > 0)
        // {
        //     hipMemcpy(result_observed_cloud_label, cuda_cloud_mask_label, rendered_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
        // }
        // // for (int i = 0; i < rendered_cloud_point_num; i++)
        // // {
        // //     printf("%d ", cloud_pose_map[i]);
        // // }
        // // printf("\n");
        // // for(int n = 0; n < num_poses; n ++)
        // // {
        // //     for(int i = 0; i < height; i ++)
        // //     {
        // //         for(int j = 0; j < width; j ++)
        // //         {
        // //             int index = n*width*height + (i*width + j);
        // //             int cloud_index = mask[index];
        // //             // printf("cloud_i:%d\n", cloud_index);
        // //             if (depth_data[index] > 0)
        // //             {
        // //                 // printf("x:%f,y:%f,z:%f\n", 
        // //                 // result_cloud[3*cloud_index], result_cloud[3*cloud_index + 1], result_cloud[3*cloud_index + 2]);
        // //             }
        // //         }
        // //     }
        // // }
        // // if (hipGetLastError() != hipSuccess) 
        // // {
        // //     printf("ERROR: Unable to execute kernel depth_to_cloud\n");
        // //     hipFree(depth_data_cuda);
        // //     hipFree(pose_occluded_cuda);
        // //     hipFree(cuda_cloud);
        // //     hipFree(cuda_cloud_color);
        // //     hipFree(cuda_cloud_pose_map);
        // //     if (label_mask_data != NULL)
        // //     {
        // //         hipFree(cuda_cloud_mask_label);
        // //     }
        // //     return false;
        // // }
        // printf("depth2cloud_global() Done\n");
        // hipFree(depth_data_cuda);
        // hipFree(pose_occluded_cuda);
        // hipFree(cuda_cloud);
        // hipFree(cuda_cloud_color);
        // hipFree(cuda_cloud_pose_map);
        // if (label_mask_data.size() > 0)
        // {
        //     hipFree(cuda_cloud_mask_label);
        // }
        // if (observed_cloud_bounds.size() > 0)
        // {
        //     hipFree(observed_cloud_bounds_cuda);
        // }
        // if (camera_transform != NULL)
        // {
        //     hipFree(camera_transform_cuda);
        // }
        // return true;
    }
    
    bool compute_rgbd_cost(
        float &sensor_resolution,
        float* knn_dist,
        int* knn_index,
        int* poses_occluded,
        int* cloud_pose_map,
        float* observed_cloud,
        uint8_t* observed_cloud_color,
        float* rendered_cloud,
        uint8_t* rendered_cloud_color,
        int rendered_cloud_point_num,
        int observed_cloud_point_num,
        int num_poses,
        float* &rendered_cost,
        std::vector<float> pose_observed_points_total,
        float* &observed_cost,
        int* pose_segmentation_label,
        int* result_observed_cloud_label,
        int cost_type,
        bool calculate_observed_cost
    )
    {
        /*
         * Function not mainted
         * @pose_observed_points_total - number of total points in observed scene corresponding to given object
         * It is calculated using segmentation label
         * @result_observed_cloud_label - label for every point in the observed cloud, used in calculating render cost,
         * A point is penalized only if it belongs to same label (pose of point and closest observed point)
         */
        // for (int i = 0; i < num_poses; i++)
        // {
        //     printf("%d ", poses_occluded[i]);
        // }
        // printf("\n");
        printf("compute_cost()\n");

        float* cuda_knn_dist;
        int* cuda_knn_index;
        // float* cuda_sensor_resolution;
        int* cuda_poses_occluded;
        int* cuda_cloud_pose_map;
        float* cuda_rendered_cost;
        float* cuda_pose_point_num;
        uint8_t* cuda_observed_cloud_color;
        uint8_t* cuda_rendered_cloud_color;
        float* cuda_rendered_cloud;
        uint8_t* cuda_observed_explained;

        int* cuda_pose_segmentation_label;
        int* cuda_observed_cloud_label;

        const unsigned int size_of_float = sizeof(float);
        const unsigned int size_of_int   = sizeof(int);
        const unsigned int size_of_uint   = sizeof(uint8_t);

        hipMalloc(&cuda_knn_dist, rendered_cloud_point_num * size_of_float);
        hipMalloc(&cuda_knn_index, rendered_cloud_point_num * size_of_int);
        hipMalloc(&cuda_cloud_pose_map, rendered_cloud_point_num * size_of_int);
        hipMalloc(&cuda_observed_cloud_color, 3 * observed_cloud_point_num * size_of_uint);
        hipMalloc(&cuda_rendered_cloud, 3 * rendered_cloud_point_num * size_of_float);
        hipMalloc(&cuda_rendered_cloud_color, 3 * rendered_cloud_point_num * size_of_uint);
        hipMalloc(&cuda_poses_occluded, num_poses * size_of_int);
        hipMalloc(&cuda_pose_segmentation_label, num_poses * size_of_int);
        hipMalloc(&cuda_observed_cloud_label, observed_cloud_point_num * size_of_int);

        thrust::device_vector<float> cuda_rendered_cost_vec(num_poses, 0);
        cuda_rendered_cost = thrust::raw_pointer_cast(cuda_rendered_cost_vec.data());
        thrust::device_vector<float> cuda_pose_point_num_vec(num_poses, 0);
        cuda_pose_point_num = thrust::raw_pointer_cast(cuda_pose_point_num_vec.data());

        thrust::device_vector<uint8_t> cuda_observed_explained_vec(num_poses * observed_cloud_point_num, 0);
        cuda_observed_explained = thrust::raw_pointer_cast(cuda_observed_explained_vec.data());

        hipMemcpy(cuda_knn_dist, knn_dist, rendered_cloud_point_num * size_of_float, hipMemcpyHostToDevice);
        hipMemcpy(cuda_knn_index, knn_index, rendered_cloud_point_num * size_of_int, hipMemcpyHostToDevice);
        hipMemcpy(cuda_cloud_pose_map, cloud_pose_map, rendered_cloud_point_num * size_of_int, hipMemcpyHostToDevice);
        hipMemcpy(cuda_observed_cloud_color, observed_cloud_color, 3 * observed_cloud_point_num * size_of_uint, hipMemcpyHostToDevice);
        hipMemcpy(cuda_rendered_cloud, rendered_cloud, 3 * rendered_cloud_point_num * size_of_float, hipMemcpyHostToDevice);
        hipMemcpy(cuda_rendered_cloud_color, rendered_cloud_color, 3 * rendered_cloud_point_num * size_of_uint, hipMemcpyHostToDevice);
        hipMemcpy(cuda_poses_occluded, poses_occluded, num_poses * size_of_int, hipMemcpyHostToDevice);
        
        hipMemcpy(cuda_pose_segmentation_label, pose_segmentation_label, num_poses * size_of_int, hipMemcpyHostToDevice);
        hipMemcpy(cuda_observed_cloud_label, result_observed_cloud_label, observed_cloud_point_num * size_of_int, hipMemcpyHostToDevice);

        const size_t threadsPerBlock = 256;
        dim3 numBlocksR((rendered_cloud_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
        compute_render_cost<<<numBlocksR, threadsPerBlock>>>(
            cuda_knn_dist,
            cuda_knn_index,
            cuda_cloud_pose_map,
            cuda_poses_occluded,
            cuda_rendered_cost,
            sensor_resolution,
            rendered_cloud_point_num,
            observed_cloud_point_num,
            cuda_pose_point_num,
            cuda_rendered_cloud_color,
            cuda_observed_cloud_color,
            cuda_rendered_cloud,
            cuda_observed_explained,
            cuda_pose_segmentation_label,
            cuda_observed_cloud_label,
            cost_type,
            15);
        

        

        if (hipGetLastError() != hipSuccess) {
            printf("ERROR: Unable to execute kernel\n");
            hipFree(cuda_knn_dist);
            hipFree(cuda_knn_index);
            hipFree(cuda_cloud_pose_map); 
            hipFree(cuda_observed_cloud_color); 
            hipFree(cuda_rendered_cloud); 
            hipFree(cuda_rendered_cloud_color); 
            hipFree(cuda_poses_occluded); 
            // hipFree(cuda_rendered_cost); 
            // hipFree(cuda_observed_explained); 
            hipFree(cuda_pose_segmentation_label);
            hipFree(cuda_observed_cloud_label);
            return false;
        }

        thrust::device_vector<float> rendered_multiplier_val(num_poses, 100);
        if (true)
        {
            thrust::transform(
                cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
                cuda_pose_point_num_vec.begin(), cuda_rendered_cost_vec.begin(), 
                thrust::divides<float>()
            );
            thrust::transform(
                cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
                rendered_multiplier_val.begin(), cuda_rendered_cost_vec.begin(), 
                thrust::multiplies<float>()
            );
        }
        rendered_cost = (float*) malloc(num_poses * size_of_float);
        hipMemcpy(rendered_cost, cuda_rendered_cost, num_poses * size_of_float, hipMemcpyDeviceToHost);


        // Compute observe cost using points marked in render cost kernel
        if (calculate_observed_cost)
        {
            thrust::device_vector<float> cuda_pose_observed_explained_vec(num_poses, 0);
            float* cuda_pose_observed_explained = thrust::raw_pointer_cast(cuda_pose_observed_explained_vec.data());

            dim3 numBlocksO((num_poses * observed_cloud_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
            compute_observed_cost<<<numBlocksO, threadsPerBlock>>>(
                num_poses,
                observed_cloud_point_num,
                cuda_observed_explained,
                cuda_pose_observed_explained
            );
            
            // Subtract total observed points for each pose with explained points for each pose
            thrust::device_vector<float> cuda_pose_observed_points_total_vec = pose_observed_points_total;
            thrust::device_vector<float> cuda_observed_cost_vec(num_poses, 0);
            thrust::transform(
                cuda_pose_observed_points_total_vec.begin(), cuda_pose_observed_points_total_vec.end(), 
                cuda_pose_observed_explained_vec.begin(), cuda_observed_cost_vec.begin(), 
                thrust::minus<float>()
            );

            // printf("Observed explained\n");
            // thrust::copy(
            //     cuda_pose_observed_points_total_vec.begin(),
            //     cuda_pose_observed_points_total_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );
            // Divide by total points
            thrust::transform(
                cuda_observed_cost_vec.begin(), cuda_observed_cost_vec.end(), 
                cuda_pose_observed_points_total_vec.begin(), cuda_observed_cost_vec.begin(), 
                thrust::divides<float>()
            );

            // Multiply by 100
            thrust::transform(
                cuda_observed_cost_vec.begin(), cuda_observed_cost_vec.end(), 
                rendered_multiplier_val.begin(), cuda_observed_cost_vec.begin(), 
                thrust::multiplies<float>()
            );

            // printf("Observed cost\n");
            // thrust::copy(
            //     cuda_observed_cost_vec.begin(),
            //     cuda_observed_cost_vec.end(), 
            //     std::ostream_iterator<int>(std::cout, " ")
            // );

            observed_cost = (float*) malloc(num_poses * size_of_float);
            float* cuda_observed_cost = thrust::raw_pointer_cast(cuda_observed_cost_vec.data());
            hipMemcpy(observed_cost, cuda_observed_cost, num_poses * size_of_float, hipMemcpyDeviceToHost);
        
        }

        // result_observed_explained = (uint8_t*) malloc(num_poses * observed_cloud_point_num * size_of_uint);
        // hipMemcpy(result_observed_explained, cuda_observed_explained, num_poses * observed_cloud_point_num * size_of_uint, hipMemcpyDeviceToHost);

        // for (int i = 0; i < num_poses; i++)
        // {
        //     printf("%f ", rendered_cost[i]);
        // }
        // printf("\n");

        printf("compute_cost() done\n");
        hipFree(cuda_knn_dist);
        hipFree(cuda_knn_index);
        hipFree(cuda_cloud_pose_map); 
        hipFree(cuda_observed_cloud_color); 
        hipFree(cuda_rendered_cloud); 
        hipFree(cuda_rendered_cloud_color); 
        hipFree(cuda_poses_occluded); 
        // hipFree(cuda_rendered_cost); 
        // hipFree(cuda_observed_explained); 
        hipFree(cuda_pose_segmentation_label);
        hipFree(cuda_observed_cloud_label);
        return true;
    }
}

