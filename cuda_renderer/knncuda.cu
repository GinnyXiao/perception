#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda_renderer/renderer.h"
#include "cuda_renderer/knncuda.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define BLOCK_DIM 16


/**
 * Computes the squared Euclidean distance matrix between the query points and the reference points.
 *
 * @param ref          refence points stored in the global memory
 * @param ref_width    number of reference points
 * @param ref_pitch    pitch of the reference points array in number of column
 * @param query        query points stored in the global memory
 * @param query_width  number of query points
 * @param query_pitch  pitch of the query points array in number of columns
 * @param height       dimension of points = height of texture `ref` and of the array `query`
 * @param dist         array containing the query_width x ref_width computed distances
 */
 namespace cuda_renderer {
__global__ void compute_distances(float * ref,
                                  int     ref_width,
                                  int     ref_pitch,
                                  float * query,
                                  int     query_width,
                                  int     query_pitch,
                                  int     height,
                                  float * dist) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Initializarion of the SSD for the current thread
    float ssd = 0.f;

    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * ref_pitch;
    step_B  = BLOCK_DIM * query_pitch;
    end_A   = begin_A + (height-1) * ref_pitch;

    // Conditions
    int cond0 = (begin_A + tx < ref_width); // used to write in shared memory
    int cond1 = (begin_B + tx < query_width); // used to write in shared memory & to computations and to write in output array 
    int cond2 = (begin_A + ty < ref_width); // used to computations and to write in output matrix

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k){
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

        // Synchronize to make sure that the preceeding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1) {
        dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
    }
}


/**
 * Computes the squared Euclidean distance matrix between the query points and the reference points.
 *
 * @param ref          refence points stored in the texture memory
 * @param ref_width    number of reference points
 * @param query        query points stored in the global memory
 * @param query_width  number of query points
 * @param query_pitch  pitch of the query points array in number of columns
 * @param height       dimension of points = height of texture `ref` and of the array `query`
 * @param dist         array containing the query_width x ref_width computed distances
 */
__global__ void compute_distance_texture(hipTextureObject_t ref,
                                         int                 ref_width,
                                         float *             query,
                                         int                 query_width,
                                         int                 query_pitch,
                                         int                 height,
                                         float*              dist) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ( xIndex<query_width && yIndex<ref_width) {
        float ssd = 0.f;
        for (int i=0; i<height; i++) {
            float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            ssd += tmp * tmp;
        }
        dist[yIndex * query_pitch + xIndex] = ssd;
    }
}


/**
 * For each reference point (i.e. each column) finds the k-th smallest distances
 * of the distance matrix and their respective indexes and gathers them at the top
 * of the 2 arrays.
 *
 * Since we only need to locate the k smallest distances, sorting the entire array
 * would not be very efficient if k is relatively small. Instead, we perform a
 * simple insertion sort by eventually inserting a given distance in the first
 * k values.
 *
 * @param dist         distance matrix
 * @param dist_pitch   pitch of the distance matrix given in number of columns
 * @param index        index matrix
 * @param index_pitch  pitch of the index matrix given in number of columns
 * @param width        width of the distance matrix and of the index matrix
 * @param height       height of the distance matrix
 * @param k            number of values to find
 */
__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){

    // Column position
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Do nothing if we are out of bounds
    if (xIndex < width) {

        // Pointer shift
        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;

        // Initialise the first index
        p_index[0] = 0;

        // Go through all points
        for (int i=1; i<height; ++i) {

            // Store current distance and associated index
            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

            // Write the current distance and index at their position
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}


/**
 * Computes the square root of the first k lines of the distance matrix.
 *
 * @param dist   distance matrix
 * @param width  width of the distance matrix
 * @param pitch  pitch of the distance matrix given in number of columns
 * @param k      number of values to consider
 */
__global__ void compute_sqrt(float * dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}


/**
 * Computes the squared norm of each column of the input array.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param height  number of rows of `array` = dimension of the points
 * @param norm    output array containing the squared norm values
 */
__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex<width){
        float sum = 0.f;
        for (int i=0; i<height; i++){
            float val = array[i*pitch+xIndex];
            sum += val*val;
        }
        norm[xIndex] = sum;
    }
}


/**
 * Add the reference points norm (column vector) to each colum of the input array.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param height  number of rows of `array` = dimension of the points
 * @param norm    reference points norm stored as a column vector
 */
__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx==0 && yIndex<height)
        shared_vec[ty] = norm[yIndex];
    __syncthreads();
    if (xIndex<width && yIndex<height)
        array[yIndex*pitch+xIndex] += shared_vec[ty];
}


/**
 * Adds the query points norm (row vector) to the k first lines of the input
 * array and computes the square root of the resulting values.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param k       number of neighbors to consider
 * @param norm     query points norm stored as a row vector
 */
__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        array[yIndex*pitch + xIndex] = sqrt(array[yIndex*pitch + xIndex] + norm[xIndex]);
}
__global__ void depth_to_mask(
    int32_t* depth, int* mask, int width, int height, int stride, int* pose_occluded)
{
    int n = (int)floorf((blockIdx.x * blockDim.x + threadIdx.x)/(width/stride));
    int x = (blockIdx.x * blockDim.x + threadIdx.x)%(width/stride);
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    x = x*stride;
    y = y*stride;
    if(x >= width) return;
    if(y >= height) return;
    uint32_t idx_depth = n * width * height + x + y*width;
    uint32_t idx_mask = n * width * height + x + y*width;

    if(depth[idx_depth] > 0 && !pose_occluded[n]) 
    {
        mask[idx_mask] = 1;
    }
}

__global__ void depth_to_cloud(
    int32_t* depth, float* cloud, int cloud_rendered_cloud_point_num, int* mask, int width, int height, 
    float kCameraCX, float kCameraCY, float kCameraFX, float kCameraFY, float depth_factor,
    int stride, int* cloud_pose_map)
{
    int n = (int)floorf((blockIdx.x * blockDim.x + threadIdx.x)/(width/stride));
    int x = (blockIdx.x * blockDim.x + threadIdx.x)%(width/stride);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // uint32_t x = blockIdx.x*blockDim.x + threadIdx.x;
    // uint32_t y = blockIdx.y*blockDim.y + threadIdx.y;
    x = x*stride;
    y = y*stride;
    if(x >= width) return;
    if(y >= height) return;
    uint32_t idx_depth = n * width * height + x + y*width;

    if(depth[idx_depth] <= 0) return;

    // printf("depth:%d\n", depth[idx_depth]);
    // uchar depth_val = depth[idx_depth];
    float z_pcd = static_cast<float>(depth[idx_depth])/depth_factor;
    float x_pcd = (static_cast<float>(x) - kCameraCX)/kCameraFX * z_pcd;
    float y_pcd = (static_cast<float>(y) - kCameraCY)/kCameraFY * z_pcd;
    // printf("kCameraCX:%f,kCameraFX:%f, kCameraCY:%f, kCameraCY:%f\n", kCameraCX,kCameraFX,kCameraCY, y_pcd, z_pcd);

    // printf("x:%d,y:%d, x_pcd:%f, y_pcd:%f, z_pcd:%f\n", x,y,x_pcd, y_pcd, z_pcd);
    uint32_t idx_mask = n * width * height + x + y*width;
    int cloud_idx = mask[idx_mask];
    cloud[cloud_idx + 0*cloud_rendered_cloud_point_num] = x_pcd;
    cloud[cloud_idx + 1*cloud_rendered_cloud_point_num] = y_pcd;
    cloud[cloud_idx + 2*cloud_rendered_cloud_point_num] = z_pcd;
    cloud_pose_map[cloud_idx] = n;
    // printf("cloud_idx:%d\n", cloud_pose_map[cloud_idx]);

    // cloud[3*cloud_idx + 0] = x_pcd;
    // cloud[3*cloud_idx + 1] = y_pcd;
    // cloud[3*cloud_idx + 2] = z_pcd;
}

bool depth2cloud_global(int32_t* depth_data,
                        float* &result_cloud,
                        int* &dc_index,
                        int &rendered_cloud_point_num,
                        int* &cloud_pose_map,
                        int width, 
                        int height, 
                        int num_poses,
                        int* pose_occluded,
                        float kCameraCX, 
                        float kCameraCY, 
                        float kCameraFX, 
                        float kCameraFY,
                        float depth_factor,
                        int stride,
                        int point_dim)
{
    printf("depth2cloud_global()\n");
    // int size = num_poses * width * height * sizeof(float);
    // int point_dim = 3;
    // int* depth_data = result_depth.data();
    // float* cuda_cloud;
    // // int* mask;

    // hipMalloc(&cuda_cloud, point_dim*size);
    // hipMalloc(&mask, size);

    int32_t* depth_data_cuda;
    int* pose_occluded_cuda;
    // int stride = 5;
    hipMalloc(&depth_data_cuda, num_poses * width * height * sizeof(int32_t));
    hipMemcpy(depth_data_cuda, depth_data, num_poses * width * height * sizeof(int32_t), hipMemcpyHostToDevice);
    
    hipMalloc(&pose_occluded_cuda, num_poses * sizeof(int));
    hipMemcpy(pose_occluded_cuda, pose_occluded, num_poses * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width/stride * num_poses + threadsPerBlock.x - 1)/threadsPerBlock.x, (height/stride + threadsPerBlock.y - 1)/threadsPerBlock.y);

    thrust::device_vector<int> mask(width*height*num_poses, 0);
    int* mask_ptr = thrust::raw_pointer_cast(mask.data());

    depth_to_mask<<<numBlocks, threadsPerBlock>>>(depth_data_cuda, mask_ptr, width, height, stride, pose_occluded_cuda);
    if (hipGetLastError() != hipSuccess) 
    {
        printf("ERROR: Unable to execute kernel\n");
        return false;
    }
    hipDeviceSynchronize();

    // Create mapping from pixel to corresponding index in point cloud
    int mask_back_temp = mask.back();
    thrust::exclusive_scan(mask.begin(), mask.end(), mask.begin(), 0); // in-place scan
    rendered_cloud_point_num = mask.back() + mask_back_temp;
    printf("Actual points in all clouds : %d\n", rendered_cloud_point_num);

    float* cuda_cloud;
    int* cuda_cloud_pose_map;
    hipMalloc(&cuda_cloud, point_dim * rendered_cloud_point_num * sizeof(float));
    hipMalloc(&cuda_cloud_pose_map, rendered_cloud_point_num * sizeof(int));

    result_cloud = (float*) malloc(point_dim * rendered_cloud_point_num * sizeof(float));
    dc_index = (int*) malloc(num_poses * width * height * sizeof(int));
    cloud_pose_map = (int*) malloc(rendered_cloud_point_num * sizeof(int));

    depth_to_cloud<<<numBlocks, threadsPerBlock>>>(
                        depth_data_cuda, cuda_cloud, rendered_cloud_point_num, mask_ptr, width, height, 
                        kCameraCX, kCameraCY, kCameraFX, kCameraFY, depth_factor, stride, cuda_cloud_pose_map);
        
    hipDeviceSynchronize();
    hipMemcpy(result_cloud, cuda_cloud, point_dim * rendered_cloud_point_num * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dc_index, mask_ptr, num_poses * width * height * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cloud_pose_map, cuda_cloud_pose_map, rendered_cloud_point_num * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i < rendered_cloud_point_num; i++)
    // {
    //     printf("%d ", cloud_pose_map[i]);
    // }
    // printf("\n");
    // for(int n = 0; n < num_poses; n ++)
    // {
    //     for(int i = 0; i < height; i ++)
    //     {
    //         for(int j = 0; j < width; j ++)
    //         {
    //             int index = n*width*height + (i*width + j);
    //             int cloud_index = mask[index];
    //             // printf("cloud_i:%d\n", cloud_index);
    //             if (depth_data[index] > 0)
    //             {
    //                 // printf("x:%f,y:%f,z:%f\n", 
    //                 // result_cloud[3*cloud_index], result_cloud[3*cloud_index + 1], result_cloud[3*cloud_index + 2]);
    //             }
    //         }
    //     }
    // }
    if (hipGetLastError() != hipSuccess) 
    {
        printf("ERROR: Unable to execute kernel\n");
        return false;
    }
    printf("depth2cloud_global() Done\n");
    hipFree(depth_data_cuda);
    hipFree(cuda_cloud);
    hipFree(pose_occluded_cuda);
    return true;
}
__global__ void compute_render_cost(
        float* cuda_knn_dist,
        int* cuda_cloud_pose_map,
        int* cuda_poses_occluded,
        float* cuda_rendered_cost,
        float sensor_resolution,
        int rendered_cloud_point_num,
        float* cuda_pose_point_num
    )
{
    size_t point_index = blockIdx.x*blockDim.x + threadIdx.x;
    if(point_index >= rendered_cloud_point_num) return;

    int pose_index = cuda_cloud_pose_map[point_index];
    if (cuda_poses_occluded[pose_index])
    {
        cuda_rendered_cost[pose_index] = -1;
    }
    else
    {
        atomicAdd(&cuda_pose_point_num[pose_index], 1);
        if (cuda_knn_dist[point_index] > sensor_resolution)
        {
            atomicAdd(&cuda_rendered_cost[pose_index], 1);
        }
    }
}
bool compute_cost(
    float &sensor_resolution,
    float* knn_dist,
    int* knn_index,
    int* poses_occluded,
    int* cloud_pose_map,
    float* result_observed_cloud,
    int rendered_cloud_point_num,
    int num_poses,
    float* &rendered_cost
)
{
    // for (int i = 0; i < num_poses; i++)
    // {
    //     printf("%d ", poses_occluded[i]);
    // }
    // printf("\n");
    printf("compute_cost()\n");

    float* cuda_knn_dist;
    // float* cuda_sensor_resolution;
    int* cuda_poses_occluded;
    int* cuda_cloud_pose_map;
    float* cuda_rendered_cost;
    float* cuda_pose_point_num;

    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    hipMalloc(&cuda_knn_dist, rendered_cloud_point_num * size_of_float);
    hipMalloc(&cuda_cloud_pose_map, rendered_cloud_point_num * size_of_int);
    hipMalloc(&cuda_poses_occluded, num_poses * size_of_int);
    thrust::device_vector<float> cuda_rendered_cost_vec(num_poses, 0);
    cuda_rendered_cost = thrust::raw_pointer_cast(cuda_rendered_cost_vec.data());
    thrust::device_vector<float> cuda_pose_point_num_vec(num_poses, 0);
    cuda_pose_point_num = thrust::raw_pointer_cast(cuda_pose_point_num_vec.data());

    hipMemcpy(cuda_knn_dist, knn_dist, rendered_cloud_point_num * size_of_float, hipMemcpyHostToDevice);
    hipMemcpy(cuda_cloud_pose_map, cloud_pose_map, rendered_cloud_point_num * size_of_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_poses_occluded, poses_occluded, num_poses * size_of_int, hipMemcpyHostToDevice);
    // hipMemcpy(cuda_sensor_resolution, &sensor_resolution, size_of_float, hipMemcpyHostToDevice);
    // hipMemset(cuda_rendered_cost, 0, num_poses * size_of_int);

    const size_t threadsPerBlock = 256;
    dim3 numBlocks((rendered_cloud_point_num + threadsPerBlock - 1) / threadsPerBlock, 1);
    compute_render_cost<<<numBlocks, threadsPerBlock>>>(
        cuda_knn_dist,
        cuda_cloud_pose_map,
        cuda_poses_occluded,
        cuda_rendered_cost,
        sensor_resolution,
        rendered_cloud_point_num,
        cuda_pose_point_num
    );

    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(cuda_knn_dist);
        hipFree(cuda_cloud_pose_map); 
        hipFree(cuda_poses_occluded); 
        hipFree(cuda_rendered_cost); 
        return false;
    }

    thrust::transform(
        cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
        cuda_pose_point_num_vec.begin(), cuda_rendered_cost_vec.begin(), 
        thrust::divides<float>()
    );
    thrust::device_vector<float> rendered_multiplier_val(num_poses, 100);
    thrust::transform(
        cuda_rendered_cost_vec.begin(), cuda_rendered_cost_vec.end(), 
        rendered_multiplier_val.begin(), cuda_rendered_cost_vec.begin(), 
        thrust::multiplies<float>()
    );
    rendered_cost = (float*) malloc(num_poses * size_of_float);
    hipMemcpy(rendered_cost, cuda_rendered_cost, num_poses * size_of_float, hipMemcpyDeviceToHost);

    // for (int i = 0; i < num_poses; i++)
    // {
    //     printf("%f ", rendered_cost[i]);
    // }
    // printf("\n");

    printf("compute_cost() done\n");
    hipFree(cuda_knn_dist);
    hipFree(cuda_cloud_pose_map); 
    hipFree(cuda_poses_occluded); 
    // hipFree(cuda_rendered_cost); 
    return true;
}
bool knn_cuda_global(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t err0, err1, err2, err3;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * ref_dev   = NULL;
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int   * index_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Compute the squared Euclidean distances
    dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
    if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
    if (ref_nb   % BLOCK_DIM != 0) grid0.y += 1;
    compute_distances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Sort the distances with their respective indexes
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Memory clean-up
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev); 

    return true;
}


bool knn_cuda_texture(const float * ref,
                      int           ref_nb,
                      const float * query,
                      int           query_nb,
                      int           dim,
                      int           k,
                      float *       knn_dist,
                      int *         knn_index) {

    // Constants
    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);   

    // Return variables
    hipError_t err0, err1, err2;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int *   index_dev = NULL;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err1 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err2 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("ERROR: Memory allocation error (hipMallocPitch)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy query data from the host to the device
    err0 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);        
        return false; 
    }

    // Allocate CUDA array for reference points
    hipArray* ref_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    err0 = hipMallocArray(&ref_array_dev, &channel_desc, ref_nb, dim);
    if (err0 != hipSuccess) {
        printf("ERROR: Memory allocation error (hipMallocArray)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        return false; 
    }

    // Copy reference points from host to device
    err0 = hipMemcpyToArray(ref_array_dev, 0, 0, ref, ref_nb * size_of_float * dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }

    // Resource descriptor
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType         = hipResourceTypeArray;
    res_desc.res.array.array = ref_array_dev;

    // Texture descriptor
    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.filterMode       = hipFilterModePoint;
    tex_desc.readMode         = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;

    // Create the texture
    hipTextureObject_t ref_tex_dev = 0;
    err0 = hipCreateTextureObject(&ref_tex_dev, &res_desc, &tex_desc, NULL);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to create the texture\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }

    // Compute the squared Euclidean distances
    dim3 block0(16, 16, 1);
    dim3 grid0(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid0.x += 1;
    if (ref_nb   % 16 != 0) grid0.y += 1;
    compute_distance_texture<<<grid0, block0>>>(ref_tex_dev, ref_nb, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Sort the distances with their respective indexes
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false; 
    }

    // Memory clean-up
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFreeArray(ref_array_dev);
    hipDestroyTextureObject(ref_tex_dev);

    return true;
}


bool knn_cublas(const float * ref,
                int           ref_nb,
                const float * query,
                int           query_nb,
                int           dim, 
                int           k, 
                float *       knn_dist,
                int *         knn_index) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t  err0, err1, err2, err3, err4, err5;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Initialize CUBLAS
    cublasInit();

    // Allocate global memory
    float * ref_dev        = NULL;
    float * query_dev      = NULL;
    float * dist_dev       = NULL;
    int   * index_dev      = NULL;
    float * ref_norm_dev   = NULL;
    float * query_norm_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    err4 = hipMalloc((void**)&ref_norm_dev,   ref_nb   * size_of_float);
    err5 = hipMalloc((void**)&query_norm_dev, query_nb * size_of_float);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Compute the squared norm of the reference points
    dim3 block0(256, 1, 1);
    dim3 grid0(ref_nb / 256, 1, 1);
    if (ref_nb % 256 != 0) grid0.x += 1;
    compute_squared_norm<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, dim, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Compute the squared norm of the query points
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    compute_squared_norm<<<grid1, block1>>>(query_dev, query_nb, query_pitch, dim, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Computation of query*transpose(reference)
    hipblasSgemm('n', 't', (int)query_pitch, (int)ref_pitch, dim, (float)-2.0, query_dev, query_pitch, ref_dev, ref_pitch, (float)0.0, dist_dev, query_pitch);
    if (cublasGetError() != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Unable to execute hipblasSgemm\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;       
    }

    // Add reference points norm
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (ref_nb   % 16 != 0) grid2.y += 1;
    add_reference_points_norm<<<grid2, block2>>>(dist_dev, query_nb, dist_pitch, ref_nb, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Sort each column
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Add query norm and compute the square root of the of the k first elements
    dim3 block3(16, 16, 1);
    dim3 grid3(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid3.x += 1;
    if (k        % 16 != 0) grid3.y += 1;
    add_query_points_norm_and_sqrt<<<grid3, block3>>>(dist_dev, query_nb, dist_pitch, k, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Memory clean-up and CUBLAS shutdown
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFree(ref_norm_dev);
    hipFree(query_norm_dev);
    cublasShutdown();

    return true;
}

    /**
     * Computes the Euclidean distance between a reference point and a query point.
     *
     * @param ref          refence points
     * @param ref_nb       number of reference points
     * @param query        query points
     * @param query_nb     number of query points
     * @param dim          dimension of points
     * @param ref_index    index to the reference point to consider
     * @param query_index  index to the query point to consider
     * @return computed distance
     */
    float compute_distance(const float * ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           ref_index,
                        int           query_index) {
        float sum = 0.f;
        for (int d=0; d<dim; ++d) {
            const float diff = ref[d * ref_nb + ref_index] - query[d * query_nb + query_index];
            sum += diff * diff;
        }
        return sqrtf(sum);
    }


    /**
     * Gathers at the beginning of the `dist` array the k smallest values and their
     * respective index (in the initial array) in the `index` array. After this call,
     * only the k-smallest distances are available. All other distances might be lost.
     *
     * Since we only need to locate the k smallest distances, sorting the entire array
     * would not be very efficient if k is relatively small. Instead, we perform a
     * simple insertion sort by eventually inserting a given distance in the first
     * k values.
     *
     * @param dist    array containing the `length` distances
     * @param index   array containing the index of the k smallest distances
     * @param length  total number of distances
     * @param k       number of smallest distances to locate
     */
    void  modified_insertion_sort_cpu(float *dist, int *index, int length, int k){

        // Initialise the first index
        index[0] = 0;

        // Go through all points
        for (int i=1; i<length; ++i) {

            // Store current distance and associated index
            float curr_dist  = dist[i];
            int   curr_index = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= dist[k-1]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = std::min(i, k-1);
            while (j > 0 && dist[j-1] > curr_dist) {
                dist[j]  = dist[j-1];
                index[j] = index[j-1];
                --j;
            }

            // Write the current distance and index at their position
            dist[j]  = curr_dist;
            index[j] = curr_index; 
        }
    }


    /*
    * For each input query point, locates the k-NN (indexes and distances) among the reference points.
    *
    * @param ref        refence points
    * @param ref_nb     number of reference points
    * @param query      query points
    * @param query_nb   number of query points
    * @param dim        dimension of points
    * @param k          number of neighbors to consider
    * @param knn_dist   output array containing the query_nb x k distances
    * @param knn_index  output array containing the query_nb x k indexes
    */
    bool knn_c(const float * ref,
            int           ref_nb,
            const float * query,
            int           query_nb,
            int           dim,
            int           k,
            float *       knn_dist,
            int *         knn_index) {

        // Allocate local array to store all the distances / indexes for a given query point 
        float * dist  = (float *) malloc(ref_nb * sizeof(float));
        int *   index = (int *)   malloc(ref_nb * sizeof(int));

        // Allocation checks
        if (!dist || !index) {
            printf("Memory allocation error\n");
            free(dist);
            free(index);
            return false;
        }

        // Process one query point at the time
        for (int i=0; i<query_nb; ++i) {

            // Compute all distances / indexes
            for (int j=0; j<ref_nb; ++j) {
                dist[j]  = compute_distance(ref, ref_nb, query, query_nb, dim, j, i);
                index[j] = j;
            }

            // Sort distances / indexes
            modified_insertion_sort_cpu(dist, index, ref_nb, k);

            // Copy k smallest distances and their associated index
            for (int j=0; j<k; ++j) {
                knn_dist[j * query_nb + i]  = dist[j];
                knn_index[j * query_nb + i] = index[j];
            }
        }

        // Memory clean-up
        free(dist);
        free(index);

        return true;

    }


    /**
     * Test an input k-NN function implementation by verifying that its output
     * results (distances and corresponding indexes) are similar to the expected
     * results (ground truth).
     *
     * Since the k-NN computation might end-up in slightly different results
     * compared to the expected one depending on the considered implementation,
     * the verification consists in making sure that the accuracy is high enough.
     *
     * The tested function is ran several times in order to have a better estimate
     * of the processing time.
     *
     * @param ref            reference points
     * @param ref_nb         number of reference points
     * @param query          query points
     * @param query_nb       number of query points
     * @param dim            dimension of reference and query points
     * @param k              number of neighbors to consider
     * @param gt_knn_dist    ground truth distances
     * @param gt_knn_index   ground truth indexes
     * @param knn            function to test
     * @param name           name of the function to test (for display purpose)
     * @param nb_iterations  number of iterations
     * return false in case of problem, true otherwise
     */
    bool knn_test(const float * ref,
            int           ref_nb,
            const float * query,
            int           query_nb,
            int           dim,
            int           k,
            float *       knn_dist,
            int *         knn_index) {

        // Parameters
        const float precision    = 0.001f; // distance error max
        const float min_accuracy = 0.999f; // percentage of correct values required
        

        // Compute the ground truth k-NN distances and indexes for each query point
        printf("Ground truth computation in progress...\n\n");
        if (!knn_c(ref, ref_nb, query, query_nb, dim, k, knn_dist, knn_index)) {
            // free(ref);
            // free(query);
            // free(knn_dist);
            // free(knn_index);
            return EXIT_FAILURE;
        }

        // Display k-NN function name
        // printf("- %-17s : ", name);

        // Allocate memory for computed k-NN neighbors
        float * test_knn_dist  = (float*) malloc(query_nb * k * sizeof(float));
        int   * test_knn_index = (int*)   malloc(query_nb * k * sizeof(int));

        // Allocation check
        if (!test_knn_dist || !test_knn_index) {
            printf("ALLOCATION ERROR\n");
            free(test_knn_dist);
            free(test_knn_index);
            return false;
        }

        // Start timer
        struct timeval tic;
        gettimeofday(&tic, NULL);

        // Compute k-NN several times
        for (int i=0; i<1; ++i) {
            if (!knn_cuda_global(ref, ref_nb, query, query_nb, dim, k, test_knn_dist, test_knn_index)) {
                free(test_knn_dist);
                free(test_knn_index);
                return false;
            }
        }

        // Stop timer
        struct timeval toc;
        gettimeofday(&toc, NULL);

        // Elapsed time in ms
        double elapsed_time = toc.tv_sec - tic.tv_sec;
        elapsed_time += (toc.tv_usec - tic.tv_usec) / 1000000.;

        // Verify both precisions and indexes of the k-NN values
        int nb_correct_precisions = 0;
        int nb_correct_indexes    = 0;
        for (int i=0; i<query_nb*k; ++i) {
            if (fabs(test_knn_dist[i] - knn_dist[i]) <= precision) {
                nb_correct_precisions++;
            }
            if (test_knn_index[i] == knn_index[i]) {
                nb_correct_indexes++;
            }
        }

        // Compute accuracy
        float precision_accuracy = nb_correct_precisions / ((float) query_nb * k);
        float index_accuracy     = nb_correct_indexes    / ((float) query_nb * k);

        // Display report
        if (precision_accuracy >= min_accuracy && index_accuracy >= min_accuracy ) {
            printf("PASSED in %8.5f seconds (averaged over %3d iterations)\n", elapsed_time / 1, 1);
        }
        else {
            printf("FAILED\n");
        }

        // Free memory
        free(test_knn_dist);
        free(test_knn_index);

        return true;
    }
}
